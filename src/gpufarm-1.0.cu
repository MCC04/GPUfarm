#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <assert.h> 
#include <cstdlib>
#include <algorithm>
#include <ctime>
#include <vector>
#include <future>
#include <iterator>

#define HIGH 500.0f
#define LOW -500.0f

struct my_struct {
    float *x_vect;
    int *clocks;
    float eventTime;
};

//#define STREAM: is the same as compile with -DSTREAM flag

//M = iterations; N = size
__global__ void cosKernel(int M, int N, float *x_d, int offset, int *myclocks){    
    int idx = offset+blockIdx.x*blockDim.x + threadIdx.x; 
    float tmp;

    clock_t start =clock();

    tmp=x_d[idx];
    for(int j=0;j<M;j+=1){
        //tmp=x_d[idx];
        //x_d[idx]=cosf(tmp);
        x_d[idx]=cosf(x_d[idx]);
    }    

    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x]=(int)(end-start);

    //myclocks[idx]=(int)(end-start);
    return ;
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
    #if defined(DEBUG) || defined(_DEBUG)
        if (result != hipSuccess) {
        std::cout <<  "CUDA Runtime Error: " << hipGetErrorString(result)<< std::endl;
        assert(result == hipSuccess);
        }
    #endif
        return result;
}

void printResults(float *cosx, int *clks){

}

int main(int argc, char **argv){
    std::srand(static_cast <unsigned> (time(NULL)));
    int blockSize=56;
    int gpu_clk=1;
    float clockSum=0.0, clockAvg=0.0;

    //if (argc > 1) devId = atoi(argv[1]);

    int devId = atoi(argv[1]);
    int K_exec = atoi(argv[2]);
    int M_iter = atoi(argv[3]);
    int N_size = atoi(argv[4]);

    const int bytesSize = N_size*sizeof(float);    
    float *x, *x_d, *cosx;
    x=new float [N_size];
    cosx=new float[N_size];  
    
   
    float ms=0.0; // elapsed time in milliseconds
    float msSum=0.0;

    checkCuda(hipDeviceGetAttribute(&gpu_clk, hipDeviceAttributeClockRate, devId));

    hipDeviceProp_t prop;
    checkCuda( hipSetDevice(devId) );
    checkCuda( hipGetDeviceProperties(&prop, devId));
    std::cout<<"Device : "<< prop.name <<std::endl;
    std::cout<<"multiproc num : "<< prop.multiProcessorCount <<std::endl;
    std::cout<<"warp size : "<< prop.warpSize <<std::endl;
    std::cout<<"GPU freq (kHz) : "<< gpu_clk <<std::endl<<std::endl;


    
    //random generation of X vector
    //std::cout<<"X array : " <<std::endl;    
   /* for(int i=0; i<N_size;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
        std::cout<< x[i] << ", ";    
    }
    std::cout <<std::endl<<"********************"<<std::endl;  
    */

std::cout << "Items number \t Host iterations \t Kernel iterations " << std::endl;
std::cout << N_size<<" \t \t \t " << K_exec<< " \t \t \t " << M_iter << std::endl;

#ifdef FUTURE
    std::cout<<std::endl<<"##########################" <<std::endl;
    std::cout<<"##########FUTURE##########" <<std::endl;
    std::cout<<"##########################" <<std::endl;

    std::vector<std::future<my_struct>> futures;
    std::vector<my_struct> getDatas;
    int GRID=N_size/blockSize;
    int *clks, *clocks_d;
    //clocks=new int[N_size]; 
    clks=new int[GRID]; 

    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );  
  

    checkCuda(hipMalloc(&x_d, bytesSize)); 
    checkCuda(hipMalloc(&clocks_d, GRID*sizeof(int)));

    for(int i=0; i<N_size;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
        //std::cout<< x[i] << ", ";    
    }

    for(int i = 0; i < K_exec; ++i) {
        futures.push_back (std::async(std::launch::deferred,
            [&]() { //int M, int N, float *x
                my_struct _xs;

                _xs.clocks=new int[GRID];
                _xs.x_vect=new float[N_size];
                checkCuda( hipEventRecord(startEvent,0) );
                checkCuda(hipMemcpy(x_d, x, bytesSize, hipMemcpyHostToDevice)); 

                cosKernel<<<GRID, blockSize>>>(M_iter, N_size, x_d, 0,clocks_d);
                
                checkCuda(hipMemcpy( _xs.x_vect, x_d, bytesSize, hipMemcpyDeviceToHost));
                checkCuda(hipMemcpy(_xs.clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost));

                checkCuda( hipEventRecord(stopEvent, 0) );
                checkCuda( hipEventSynchronize(stopEvent) );
                checkCuda( hipEventElapsedTime(&_xs.eventTime, startEvent, stopEvent) );
                            
                
                //_xs.x_vect=cosx;
                //_xs.clocks=clks;
                //_xs.eventTime=ms;
                return _xs;
            }));          
    }
    for(auto &e : futures) 
        getDatas.push_back(e.get());

    int count=0;
    for(auto item : getDatas){   
        std::cout<< std::endl << "********** ITERATION "<<count<<" **********"<< std::endl;     
        std::cout<< std::endl << "####### COSX vector: "<< std::endl;
        for(int j=0; j<N_size;j+=1) {
            std::cout << item.x_vect[j] << ", ";
        }
        std::cout<< "Clock measures"<< std::endl;
        clockSum=0;

        int max=item.clocks[0],min=item.clocks[0];
        for(int j=0; j<GRID;j+=1) {
            std::cout<< item.clocks[j] << ", ";
            clockSum+=item.clocks[j];
            if(item.clocks[j]<min) min=item.clocks[j];
            if(item.clocks[j]>max) max=item.clocks[j];
        }
        clockAvg=clockSum/GRID; 
        //auto minmax = std::minmax_element(std::begin(item.clocks), std::end(item.clocks));
        std::cout<< std::endl<<"-------------------------"<< std::endl; 
        std::cout<< "Avg clk (ms) \t min clk \t max clk \t event time (ms) "<< std::endl;   
        std::cout << clockAvg/(float)gpu_clk << " \t "<< min << " \t \t " << max << " \t \t "<< item.eventTime <<std::endl; 
       // std::cout<< std::endl "GPU freq:"<<gpu_clk <<"kHz"<<std::endl<< "Average clocks in millisec: "<< clockAvg/(float)gpu_clk << "ms"  << std::endl;        
        //std::cout << "min clk " << *(minmax.first) << std::endl << "max clk " << *(minmax.second) << std::endl;
        //std::cout<< std::endl << "Total clocks in millisec (approx): "<< clockSum/(float)gpu_clk << "ms"<<std::endl; 
   
        count+=1;
        msSum+=item.eventTime;

    } 

    float rb_wb=K_exec*(bytesSize*2 + GRID*sizeof(float));
    std::cout<<std::endl<<"----Effective Bandwidth: "<< (rb_wb/msSum/1e6)<<"GB/s"<<std::endl;

#elif STREAM
    std::cout<<std::endl<<"##########################" <<std::endl;
    std::cout<<"##########STREAM##########" <<std::endl;
    std::cout<<"##########################" <<std::endl;
    const int streamSize = N_size / K_exec ;
    const int streamBytes = streamSize* sizeof(float) ;
    //const int streamBytesInt = streamSize* sizeof(int) ;
    int GRID=streamSize/blockSize;
    int *clocks, *clocks_d;
    clocks=new int[GRID]; 

    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );

    std::cout << "Stream Size \t Stream bytes \t GRID \t BLOCK " << std::endl;
    std::cout << streamSize<<" \t \t " <<streamBytes<< " \t \t " << GRID <<" \t " << blockSize << std::endl;

    //allocate Unified Memory
    /*hipMallocManaged(&x_d, bytesSize);
    hipMallocManaged(&x, bytesSize);
    hipMallocManaged(&clocks, bytesSize);
    hipMallocManaged(&clocks_d, bytesSize);*/

    hipMalloc(&x_d, bytesSize);
    hipMalloc(&clocks_d, GRID*sizeof(int));

    //streams creation
    hipStream_t stream[K_exec];
    for (int i = 0; i < K_exec; ++i)
        checkCuda(hipStreamCreate(&stream[i]));

    memset(x, 0, bytesSize);
    memset(cosx, 0, bytesSize);
    memset(clocks, 0, GRID*sizeof(int));
       
        //random generation of X vector
    //std::cout<<"X array : " <<std::endl;    
    for(int i=0; i<N_size;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
       // std::cout<< x[i] << ", ";    
    }

    for (int r = 0; r < K_exec; ++r) {
        checkCuda( hipEventRecord(startEvent,0) );

        for (int i = 0; i < K_exec; ++i) {
            int offset = i * streamSize;
            hipMemcpyAsync(&x_d[offset], &x[offset], streamBytes, hipMemcpyHostToDevice, stream[i]);    

            cosKernel<<<GRID, blockSize, 0, stream[i]>>>(M_iter, N_size, x_d, offset, clocks_d);
            //hipDeviceSynchronize();

            hipMemcpyAsync( &cosx[offset], &x_d[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]);
            //hipMemcpyAsync( &clocks[offset], &clocks_d[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]);
            hipMemcpyAsync( &clocks[i], &clocks_d[i], GRID*sizeof(int), hipMemcpyDeviceToHost, stream[i]);

        }

        checkCuda( hipEventRecord(stopEvent, 0) );
        checkCuda( hipEventSynchronize(stopEvent) );
        checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
        
        std::cout<<"COSX array : " <<std::endl;  
        for(int j=0; j<N_size;j+=1) 
            std::cout << cosx[j] << ", ";    
        std::cout << std::endl;
        std::cout<< std::endl << "********** ITERATION "<<r<<" **********"<< std::endl;     

        int max=clocks[0],min=clocks[0];
        std::cout <<"Clocks measures"<< std::endl;
        for(int j=0; j<GRID;j+=1) {
            std::cout << clocks[j] << ", ";
            clockSum+=clocks[j];
            if(clocks[j]<min) min=clocks[j];
            if(clocks[j]>max) max=clocks[j];
        }
        clockAvg=clockSum/GRID;
        //auto minmax = std::minmax_element(std::begin(*(clocks)), std::end(*(clocks)));   
        std::cout<< std::endl <<"-------------------------"<< std::endl; 
        std::cout << "GPU freq (kHz) \t Avg clk (ms) \t min clk \t max clk \t event time(ms) "<< std::endl;   
        //std::cout << gpu_clk << " \t " << clockAvg/(float)gpu_clk << " \t "<< *(minmax.first) << " \t " << *(minmax.second) <<std::endl;         
        std::cout << gpu_clk << " \t " << clockAvg/(float)gpu_clk << " \t "<< min << " \t \t " << max <<" \t \t "<< ms<<std::endl;         

        //std::cout<< std::endl << "Total clocks in millisec (approx): "<< clockSum/(float)gpu_clk << "ms"<<std::endl;   
        msSum+=ms;
    }
    //streams destroy
    for (int i = 0; i < K_exec; ++i)
        checkCuda(hipStreamDestroy(stream[i]));

        float rb_wb=K_exec*(streamBytes*2 + GRID*sizeof(float));
        std::cout<<std::endl<<"----Effective Bandwidth: "<< (rb_wb/msSum/1e6)<<"GB/s"<<std::endl;


#elif STREAMMANAGED
std::cout<<std::endl<<"##################################" <<std::endl;
    std::cout<<"##########STREAM MANAGED##########" <<std::endl;
    std::cout<<"##################################" <<std::endl;
    const int streamSize = N_size / K_exec ;
    const int streamBytes = streamSize* sizeof(float) ;
    //const int streamBytesInt = streamSize* sizeof(int) ;
    int GRID=streamSize/blockSize;
    int *clocks, *clocks_d;
    clocks=new int[GRID]; 
    x_d=new float [N_size];
    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );

    std::cout << "Stream Size \t Stream bytes \t GRID \t BLOCK " << std::endl;
    std::cout << streamSize<<" \t \t " <<streamBytes<< " \t \t " << GRID <<" \t " << blockSize << std::endl;

    //allocate Unified Memory
    /*hipMallocManaged(&x_d, bytesSize);
    hipMallocManaged(&x, bytesSize);
    hipMallocManaged(&clocks, bytesSize);
    hipMallocManaged(&clocks_d, bytesSize);*/



    //streams creation
    hipStream_t stream[K_exec];
    for (int i = 0; i < K_exec; ++i)
        checkCuda(hipStreamCreate(&stream[i]));

    hipMallocManaged(&x, bytesSize);
    hipMallocManaged(&clocks, GRID*sizeof(int));

    memset(x, 0, bytesSize);
    memset(clocks, 0, GRID*sizeof(int));

        //random generation of X vector
    //std::cout<<"X array : " <<std::endl;    
    for(int i=0; i<N_size;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
        //x_d[i]=x[i];//std::cout<< x[i] << ", ";    
    }

 
       
    

    for (int r = 0; r < K_exec; ++r) {

        checkCuda( hipEventRecord(startEvent,0) );

        for (int i = 0; i < K_exec; ++i) {
            int offset = i * streamSize;
            //hipStreamAttachMemAsync(stream[i], &x[offset], 0, hipMemAttachSingle);    
            //hipStreamAttachMemAsync(stream[i],&clocks[i], 0, hipMemAttachSingle);

            hipStreamAttachMemAsync(stream[i], &x[offset], 0, hipMemAttachSingle);    
            hipStreamAttachMemAsync(stream[i],&clocks[i], 0, hipMemAttachSingle);
            cosKernel<<<GRID, blockSize, 0, stream[i]>>>(M_iter, N_size, x, offset, clocks);
            //hipDeviceSynchronize();

            //hipStreamAttachMemAsync( &x[offset], &x_d[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]);
        }

        checkCuda( hipEventRecord(stopEvent, 0) );
        checkCuda( hipEventSynchronize(stopEvent) );
        checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
        
        std::cout<< std::endl << "********** ITERATION "<<r<<" **********"<< std::endl;     
        //std::cout<<"X array : " <<std::endl;  
        /*for(int j=0; j<N_size;j+=1) {
            std::cout << x[j] << ", ";
            //x[j]=x_d[j];
        }*/
                
        //std::cout << std::endl;

        //std::copy ( x_d, x_d+N_size, std::back_inserter(*x));
        
        int max=clocks[0],min=clocks[0];
        std::cout <<"Clocks measures"<< std::endl;
        for(int j=0; j<GRID;j+=1) {
            std::cout << clocks[j] << ", ";
            clockSum+=clocks[j];
            if(clocks[j]<min) min=clocks[j];
            if(clocks[j]>max) max=clocks[j];
        }
        clockAvg=clockSum/GRID; 
        std::cout<< std::endl <<"-------------------------"<< std::endl; 
        //std::cout<< std::endl << "Average clocks in millisec: "<< clockAvg/(float)gpu_clk << "ms, GPU freq:"<<gpu_clk <<"kHz"<<std::endl;         
        //std::cout<< std::endl << "Total clocks in millisec (approx): "<< clockSum/(float)gpu_clk << "ms"<<std::endl;   
        //auto minmax = std::minmax_element(std::begin(clocks), std::end(clocks));   
        std::cout << "Avg clk (ms) \t min clk \t max clk \t event time (ms)"<< std::endl;   
        std::cout << clockAvg/(float)gpu_clk << " \t "<< min << " \t \t " << max <<" \t \t "<< ms  <<std::endl;       
        msSum+=ms;
    }
    //streams destroy
    for (int i = 0; i < K_exec; ++i)
        checkCuda(hipStreamDestroy(stream[i]));

        float rb_wb=K_exec*(streamBytes*2 + GRID*sizeof(float));
        std::cout<<std::endl<<"----Effective Bandwidth: "<< (rb_wb/msSum/1e6)<<"GB/s"<<std::endl;

#endif

    std::cout<<std::endl<<"----Total Events measures: "<< msSum<<"ms"<<std::endl;


    hipFree(x_d);
    hipFree(clocks_d);

    return 0;
}