#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <assert.h> 
#include <cstdlib>
#include <algorithm>
#include <ctime>
#include <vector>
#include <future>
#include <iterator>
#include <cudaUtils.h>
#include <imageMatrix.h>

#define HIGH 500.0f
#define LOW -500.0f

template<typename T> T getMatrixVal(T *mat, int row, int col, int width)
{
    return mat[row*width + col];
}

template<typename T> void setMatrixVal(T *mat, int row, int col, int width, T val)
{   
    mat[row*width + col] = val;
}

void randomMatrix(const int m, int n,float *mat){
    #ifndef MEASURES
       // std::cout<< "MATRIX M: "<<std::endl;  
    #endif

    for(int r = 0; r<m; ++r){
        for(int c = 0; c<n; ++c){
            float val=LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;
            setMatrixVal(mat, r, c, n, val);

            #ifndef MEASURES
              //  std::cout<< getMatrixVal<float>(mat,r,c,n) << ", ";              
            #endif
        }
        #ifndef MEASURES
          //  std::cout<< std::endl;  
        #endif
    }       
}

float* getGaussian(int dim, float sigma)
{
    float *ker=new float[dim*dim];
    float sum=0.0;
    int i,j;

    for (i=0 ; i<dim ; i++) {
        for (j=0 ; j<dim ; j++) {
            float val = exp((float)(-(i*i+j*j))/(2*sigma*sigma))
                        /(2*M_PI*sigma*sigma);
            setMatrixVal<float>(ker,i,j,dim, val);
            sum += val;
        }
    }

    for (i=0 ; i<dim ; i++) {
        for (j=0 ; j<dim ; j++) {
            float val=getMatrixVal<float>(ker,i,j,dim)/sum;
            setMatrixVal<float>(ker,i,j,dim, val);
        }
    }

    return ker;
}

/*********
**KERNELS*
**********/
__global__ void matMulKernel(float* A, float* B, float* C, int m, int k, int n, int chunk)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;


    for (int s = 0; s < chunk; ++s)
    {
        float *tmpA = &A[s*m*k];
        float *tmpB = &B[s*k*n];
        float *tmpC = &C[s*m*n];
        float tmpSum = 0;

        if(row<m && col<n) {
           // float sum = 0;
            for(int j=0;j<k;j++) {
                tmpSum += tmpA[row*k+j] * tmpB[j*n+col];
            }
            tmpC[row*n+col] = tmpSum;
        }
    }
    return ;
}

__global__ void matMulGridStride(float* A, float* B, float* C, int m, int k, int n, int chunk)//(int M, int N, float *x_d, int *myclocks, int offset){    
{
    int indexRow = blockIdx.x*blockDim.x + threadIdx.x;
    int strideRow = blockDim.x*gridDim.x;

    int indexCol = blockIdx.y*blockDim.y + threadIdx.y;
    int strideCol = blockDim.y*gridDim.y;

    /* A [M x K]
    *  B [K x N]
    *  C [M x N]
    */
    for (int s = 0; s < chunk; ++s)
    {
        float *tmpA = &A[s*m*k];
        float *tmpB = &B[s*k*n];
        float *tmpC = &C[s*m*n];
        float tmpSum = 0;

        for (int i = indexRow; i < m; i += strideRow) //M
        {
            for (int j = indexCol; j < n; j += strideCol) //N
            {
                //float sum = 0;
                for(int l=0; l<k; l++) //K
                {
                    tmpSum += tmpA[i*k + l] * tmpB[l*n + j];
                }
                tmpC[i*n + j] = tmpSum;
            }
        }
    }
    return ;
}

__global__ void squareMatMulKernel(float* A, float* B, float* C, int N, int chunk) {

   int ROW = blockIdx.y*blockDim.y+threadIdx.y;
        int COL = blockIdx.x*blockDim.x+threadIdx.x;
 
    if (ROW < N && COL < N) {

    for (int s = 0; s < chunk; s++)
    {
        float *tmpA = &A[s*N*N];
        float *tmpB = &B[s*N*N];
        float *tmpC = &C[s*N*N];
        float tmpSum = 0;
    
            for (int i = 0; i < N; i++) {
                tmpSum += tmpA[ROW * N + i] * tmpB[i * N + COL];
            }
       
        
        tmpC[ROW * N + COL] = tmpSum;
        __syncthreads();
    }


 }

/*

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;
    if (ROW < N && COL < N) {
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
        C[ROW * N + COL] = tmpSum;
        
    }*/

    return ;
}

__global__ void squareMatMulGridStrideKer(float* A, float* B, float* C, int N, int chunk) {

/*    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int Rstride = blockDim.y*gridDim.y;

    int COL = blockIdx.x*blockDim.x+threadIdx.x;
    int Cstride = blockDim.x*gridDim.x;

    

    for (int s = 0; s < chunk; s++)
    {
        float *tmpA = &A[s*N*N];
        float *tmpB = &B[s*N*N];
        float *tmpC = &C[s*N*N];
        float tmpSum = 0;

      for (int j = COL; j < N; j+=Cstride) { 
           for (int i = ROW; i < N; i+=Rstride) {

            //// if (ROW < N && COL < N) {
            //// each thread computes one element of the block sub-matrix
            ////for (int i = 0; i < N; i++) {
            
                //tmpSum += A[i * N + j] * B[j * N + i];
                tmpC[j * N + i] += tmpA[i * N + j] * tmpB[j * N + i];
            }
            ////}
            //tmpC[i * N + COL] = tmpSum;
            //tmpC[i * N + COL] = tmpSum;
        }
        //tmpC[ROW * N + COL] = tmpSum;
    }
*/
    

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    int Rstride = blockDim.y*gridDim.y;
    int Cstride = blockDim.x*gridDim.x;



    //float tmpSum = 0;

    for (int s = 0; s < chunk; s++)
    {
        float *tmpA = &A[s*N*N];
        float *tmpB = &B[s*N*N];
        float *tmpC = &C[s*N*N];
        float tmpSum = 0;

        //if (ROW < N && COL < N) {
        for (int j = COL; j < N; j+=Cstride) { 
            
           for (int k = ROW; k < N; k+=Rstride) {
               tmpSum=0;
                for (int i = 0; i < N; i++) {
                    tmpSum += tmpA[k * N + i] * tmpB[i * N + j];
                }
                tmpC[k * N + j] = tmpSum;
            }   
            
        }

    }

    
    return ;
}




__global__ void blurBoxFilterKer(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 5; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
        }
}

__global__ void gaussianBlurKer(
 const unsigned char* const inputChannel,
unsigned char* outputChannel,
int numRows, int numCols,
const float* filter, const int filterWidth)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= numCols || y >= numRows)
        return;
    int idx = y * numCols + x;
    float blur = 0.0f;
    for (int i = 0; i < filterWidth; i++) {
        for (int j = 0; j < filterWidth; j++) {
            int p_x = x + i - filterWidth/2;
            int p_y = y + j - filterWidth/2;
            p_x = min(max(p_x, 0), numCols - 1);
            p_y = min(max(p_y, 0), numRows - 1);
            float filter_value = filter[i * filterWidth + j];
            blur += filter_value *
            static_cast<float>(inputChannel[p_y * numCols + p_x]);
        }
    }
    outputChannel[idx] = blur;
    return ;
}



__global__ void blurBoxGridStride(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x;

    //int x = offset % width;
    //int y = (offset-x)/width;
    int fsize = 5; // Filter size
    //if(offset < width*height) {

    for(int i=offset; i<width*height; i+=stride)
    {    
        int x = offset % width;
        int y = (offset-x)/width;

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = ( i +ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[i *3] = output_red/hits;
        output_image[i *3+1] = output_green/hits;
        output_image[i *3+2] = output_blue/hits;
    }
    return ;
}

__global__ void gaussianBlurGridStride(
 const unsigned char* const inputChannel,
unsigned char* outputChannel,
int numRows, int numCols,
const float* filter, const int filterWidth)
{
    const unsigned int indexX = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int indexY = blockDim.y * blockIdx.y + threadIdx.y;

    const unsigned int strideX = blockDim.x * gridDim.x;
    const unsigned int strideY = blockDim.y * gridDim.y;

    //if (x >= numCols || y >= numRows)
      //  return;

    for(int k=indexX; k<numCols; k+=strideX)
    {
        for(int l=indexY; l<numRows; l+=strideY)
        {
            int idx = l * numCols + k;
            float blur = 0.0f;
            for (int i = 0; i < filterWidth; i++) {
                for (int j = 0; j < filterWidth; j++) {
                    int p_x = k + i - filterWidth/2;
                    int p_y = l + j - filterWidth/2;
                    p_x = min(max(p_x, 0), numCols - 1);
                    p_y = min(max(p_y, 0), numRows - 1);
                    float filter_value = filter[i * filterWidth + j];
                    blur += filter_value *
                    static_cast<float>(inputChannel[p_y * numCols + p_x]);
                }
            }
            outputChannel[idx] = blur;
        }
    }
    return ;
}


/*******************
**KERNEL LAUNCHERS**
********************/

//MAT MUL
float newMatMulKer(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, 
        int m, int k, int n, int chunk, hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{
    float ms=0;
    int bytesA = m*k*sizeof(float);
    int bytesB = k*n*sizeof(float);
    int bytesC = m*n*sizeof(float);

    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    hipMemcpyAsync(Ad, A, bytesA*chunk, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB*chunk, hipMemcpyHostToDevice, strm);   

    #ifdef LOWPAR
        matMulGridStride<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n, chunk);
    #else
        matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n, chunk);
    #endif

    hipMemcpyAsync( C, Cd, bytesC*chunk, hipMemcpyDeviceToHost, strm);

    return ms;
}

//SQUARE MATMUL
float newSquareMatMulKer(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, 
            int n, int chunk, hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{
    float ms=0;
    int size=n*n;
    int bytesMat=size*sizeof(float);

    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    checkCuda( hipMemcpyAsync(Ad, A, bytesMat*chunk, hipMemcpyHostToDevice, strm) );    
    checkCuda( hipMemcpyAsync(Bd, B, bytesMat*chunk, hipMemcpyHostToDevice, strm) );   

    #ifdef LOWPAR
        squareMatMulGridStrideKer<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n, chunk);
    #else
        squareMatMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n, chunk);
    #endif

    checkCuda( hipMemcpyAsync( C, Cd, bytesMat*chunk, hipMemcpyDeviceToHost, strm) );
    
    return ms;
}




















float smallSquareMatMulKer(
    float *Ad, float *Bd, float *Cd, float *C,
    int n, 
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{

    float ms;
    int bytesA=n*n*sizeof(float);
    int bytesB=n*n*sizeof(float);
    int bytesC=n*n*sizeof(float);
    float  *A=(float*)calloc(1,bytesA);//new float[M*K];
    float *B=(float*)calloc(1,bytesB);//new float[K*N] ;

    randomMatrix(n,n, A);
    randomMatrix(n,n, B);     

    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    checkCuda( hipEventRecord(start,0) );

    hipMemcpyAsync(Ad, A, bytesA, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB, hipMemcpyHostToDevice, strm);   

    #ifdef LOWPAR
        squareMatMulGridStrideKer<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n, 1);
    #else
        squareMatMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n, 1);
    #endif

    hipMemcpyAsync( C, Cd, bytesC, hipMemcpyDeviceToHost, strm);

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    free(A);
    free(B);

    return ms;
}


float squareMatMulKer(
    float *Ad, float *Bd, float *Cd, 
    int n,
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{
    float ms;    

    randomMatrix(n,n, Ad);
    randomMatrix(n,n, Bd);  
  
    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 


    checkCuda( hipEventRecord(start,0) );


    #ifdef LOWPAR
        squareMatMulGridStrideKer<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n, 1);
    #else
        squareMatMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n, 1);
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    return ms;
}




/*

float smallMatMulKer(
    float *Ad, float *Bd, float *Cd, float *C,
    int m, int k, int n, 
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{

    float ms;
    int bytesA=m*k*sizeof(float);
    int bytesB=k*n*sizeof(float);
    int bytesC=m*n*sizeof(float);
    float  *A=(float*)calloc(1,bytesA);//new float[M*K];
    float *B=(float*)calloc(1,bytesB);//new float[K*N] ;

    randomMatrix(m,k, A);
    randomMatrix(k,n, B);     

    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    checkCuda( hipEventRecord(start,0) );

    hipMemcpyAsync(Ad, A, bytesA, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB, hipMemcpyHostToDevice, strm);   

    #ifdef LOWPAR
        matMulGridStride<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #else
        matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #endif

    hipMemcpyAsync( C, Cd, bytesC, hipMemcpyDeviceToHost, strm);

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    free(A);
    free(B);

    return ms;
}


float matMulKer(
    float *Ad, float *Bd, float *Cd, 
    int m, int k, int n, 
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{
    float ms;    

    randomMatrix(m,k, Ad);
    randomMatrix(k,n, Bd);     
    
    //#ifdef LOWPAR
    //    dim3 dimBlock(4,4,1);
    //    dim3 dimGrid(1,1,1); 
    //#else
        dim3 dimBlock(BLOCK,BLOCK,1);
        dim3 dimGrid(GRIDx, GRIDy,1); 
   // #endif

    checkCuda( hipEventRecord(start,0) );


    #ifdef LOWPAR
        matMulGridStride<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #else
        matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    return ms;
}

*/






/*float smallMatMulKer(
    float *Ad, float *Bd, float *Cd, float *C,
    int m, int k, int n, 
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{

    float ms;
    int bytesA=m*k*sizeof(float);
    int bytesB=k*n*sizeof(float);
    int bytesC=m*n*sizeof(float);
    float  *A=(float*)calloc(1,bytesA);//new float[M*K];
    float *B=(float*)calloc(1,bytesB);//new float[K*N] ;

    randomMatrix(m,k, A);
    randomMatrix(k,n, B);     

    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    checkCuda( hipEventRecord(start,0) );

    hipMemcpyAsync(Ad, A, bytesA, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB, hipMemcpyHostToDevice, strm);   

    #ifdef LOWPAR
        matMulGridStride<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #else
        matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #endif

    hipMemcpyAsync( C, Cd, bytesC, hipMemcpyDeviceToHost, strm);

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    free(A);
    free(B);

    return ms;
}*/











float blurBoxFilter (
    unsigned char *img_in, unsigned char *img_out,
    int width, int height,
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{    
    float ms=0;
    //int bytes=width*height*3*sizeof(unsigned char);
       
    #ifdef LOWPAR
        dim3 blockDims(2,1,1);
        dim3 gridDims(1,1,1); 
    #else
        dim3 blockDims(BLOCK,1,1);
        dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );
    #endif
    checkCuda( hipEventRecord(start,0) ); 

    #ifdef LOWPAR
        blurBoxGridStride<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, width, height); 
    #else
        blurBoxFilterKer<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, width, height); 
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
 
    return ms;
}


float blurGaussianfilter (
    unsigned char *img_in, unsigned char *img_out,
    int width, int height,int kerdim, float sigma,
    hipStream_t strm,
    hipEvent_t start, hipEvent_t stop)
{    

    float *ker;
    int bytes=kerdim*kerdim*sizeof(unsigned char);
    float ms=0;

    checkCuda(hipMallocManaged(&ker, bytes));
    ker=getGaussian(kerdim, sigma);        
       
    #ifdef LOWPAR
        dim3 blockDims(2,2,1);
        dim3 gridDims(1,1, 1 );
    #else
        dim3 blockDims(BLOCK,BLOCK,1);        
        dim3 gridDims((width*3)/blockDims.x, (height*3)/blockDims.y, 1 ); //dim3 gridDims((unsigned int) ceil((double)(bytes/blockDims.x)), 1, 1 );
    #endif
    checkCuda( hipEventRecord(start,0) ); 

    #ifdef LOWPAR
        gaussianBlurGridStride<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, height, width, ker, kerdim);
    #else
        gaussianBlurKer<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, height, width, ker, kerdim);
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
 
    return ms;
}