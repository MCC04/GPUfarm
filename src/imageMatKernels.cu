#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <assert.h> 
#include <cstdlib>
#include <algorithm>
#include <ctime>
#include <vector>
#include <future>
#include <iterator>
#include <cudaUtils.h>
#include <imageMatrix.h>

#define HIGH 500.0f
#define LOW -500.0f

template<typename T> T getMatrixVal(T *mat, int row, int col, int width)
{
    return mat[row*width + col];
}

template<typename T> void setMatrixVal(T *mat, int row, int col, int width, T val)
{   
    mat[row*width + col] = val;
}

void randomMatrix(const int m, int n,float *mat){
    #ifndef MEASURES
        std::cout<< "MATRIX M: "<<std::endl;  
    #endif

    for(int r = 0; r<m; ++r){
        for(int c = 0; c<n; ++c){
            float val=LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;
            setMatrixVal(mat, r, c, n, val);

            #ifndef MEASURES
                std::cout<< getMatrixVal<float>(mat,r,c,n) << ", ";              
            #endif
        }
        #ifndef MEASURES
            std::cout<< std::endl;  
        #endif
    }       
}

float* getGaussian(int dim, float sigma)
{
    float *ker=new float[dim*dim];
    float sum=0.0;
    int i,j;

    for (i=0 ; i<dim ; i++) {
        for (j=0 ; j<dim ; j++) {
            float val = exp((float)(-(i*i+j*j))/(2*sigma*sigma))
                        /(2*M_PI*sigma*sigma);
            setMatrixVal<float>(ker,i,j,dim, val);
            sum += val;
        }
    }

    for (i=0 ; i<dim ; i++) {
        for (j=0 ; j<dim ; j++) {
            float val=getMatrixVal<float>(ker,i,j,dim)/sum;
            setMatrixVal<float>(ker,i,j,dim, val);
        }
    }

    return ker;
}

/*********
**KERNELS*
**********/
__global__ void matMulKernel(float* Ad, float* Bd, float* Cd, int m, int k, int n)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if(row<m && col<n) {
        float sum = 0;
        for(int j=0;j<k;j++) {
            sum += Ad[row*k+j] * Bd[j*n+col];
        }
        Cd[row*n+col] = sum;
    }
}

__global__ void matMulGridStride(float* Ad, float* Bd, float* Cd, int m, int k, int n)//(int M, int N, float *x_d, int *myclocks, int offset){    
{
    int indexRow = blockIdx.x*blockDim.x + threadIdx.x;
    int strideRow = blockDim.x*gridDim.x;

    int indexCol = blockIdx.y*blockDim.y + threadIdx.y;
    int strideCol = blockDim.y*gridDim.y;

    /* A [M x K]
    *  B [K x N]
    *  C [M x N]
    */


    for (int i = indexRow; i < m; i += strideRow) //M
    {
        for (int j = indexCol; j < n; j += strideCol) //N
        {
            float sum = 0;
            for(int l=0; l<k; l++) //K
            {
                sum += Ad[i*k + l] * Bd[l*n + j];
            }
            Cd[i*n + j] = sum;
        }
    }

    return ;
}

__global__ void squareMatMulKer(float* A, float* B, float* C, int N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}

__global__ void squareMatMulGridStrideKer(float* A, float* B, float* C, int N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int Rstride = blockDim.y*gridDim.y;

    int COL = blockIdx.x*blockDim.x+threadIdx.x;
    int Cstride = blockDim.x*gridDim.x;

    float tmpSum = 0;

    for (int i = ROW; i < N; i+=Rstride) {

        // if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        //for (int i = 0; i < N; i++) {
        for (int j = COL; i < N; i+=Cstride) {
            tmpSum += A[i * N + j] * B[j * N + i];
        }
        //}
        C[i * N + COL] = tmpSum;
    }
    
    return ;
}


__global__ void blurBoxFilterKer(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 5; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
        }
}

__global__ void gaussianBlurKer(
 const unsigned char* const inputChannel,
unsigned char* outputChannel,
int numRows, int numCols,
const float* filter, const int filterWidth)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= numCols || y >= numRows)
        return;
    int idx = y * numCols + x;
    float blur = 0.0f;
    for (int i = 0; i < filterWidth; i++) {
        for (int j = 0; j < filterWidth; j++) {
            int p_x = x + i - filterWidth/2;
            int p_y = y + j - filterWidth/2;
            p_x = min(max(p_x, 0), numCols - 1);
            p_y = min(max(p_y, 0), numRows - 1);
            float filter_value = filter[i * filterWidth + j];
            blur += filter_value *
            static_cast<float>(inputChannel[p_y * numCols + p_x]);
        }
    }
    outputChannel[idx] = blur;
    return ;
}



__global__ void blurBoxGridStride(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x;

    //int x = offset % width;
    //int y = (offset-x)/width;
    int fsize = 5; // Filter size
    //if(offset < width*height) {

    for(int i=offset; i<width*height; i+=stride)
    {    
        int x = offset % width;
        int y = (offset-x)/width;

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = ( i +ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[i *3] = output_red/hits;
        output_image[i *3+1] = output_green/hits;
        output_image[i *3+2] = output_blue/hits;
    }
    return ;
}

__global__ void gaussianBlurGridStride(
 const unsigned char* const inputChannel,
unsigned char* outputChannel,
int numRows, int numCols,
const float* filter, const int filterWidth)
{
    const unsigned int indexX = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int indexY = blockDim.y * blockIdx.y + threadIdx.y;

    const unsigned int strideX = blockDim.x * gridDim.x;
    const unsigned int strideY = blockDim.y * gridDim.y;

    //if (x >= numCols || y >= numRows)
      //  return;

    for(int k=indexX; k<numCols; k+=strideX)
    {
        for(int l=indexY; l<numRows; l+=strideY)
        {
            int idx = l * numCols + k;
            float blur = 0.0f;
            for (int i = 0; i < filterWidth; i++) {
                for (int j = 0; j < filterWidth; j++) {
                    int p_x = k + i - filterWidth/2;
                    int p_y = l + j - filterWidth/2;
                    p_x = min(max(p_x, 0), numCols - 1);
                    p_y = min(max(p_y, 0), numRows - 1);
                    float filter_value = filter[i * filterWidth + j];
                    blur += filter_value *
                    static_cast<float>(inputChannel[p_y * numCols + p_x]);
                }
            }
            outputChannel[idx] = blur;
        }
    }
    return ;
}


/*******************
**KERNEL LAUNCHERS**
********************/

float smallSquareMatMulKer(
    float *Ad, float *Bd, float *Cd, float *C,
    int n, 
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{

    float ms;
    int bytesA=n*n*sizeof(float);
    int bytesB=n*n*sizeof(float);
    int bytesC=n*n*sizeof(float);
    float  *A=(float*)calloc(1,bytesA);//new float[M*K];
    float *B=(float*)calloc(1,bytesB);//new float[K*N] ;

    randomMatrix(n,n, A);
    randomMatrix(n,n, B);     

    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    checkCuda( hipEventRecord(start,0) );

    hipMemcpyAsync(Ad, A, bytesA, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB, hipMemcpyHostToDevice, strm);   

    #ifdef LOWPAR
        squareMatMulGridStrideKer<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    #else
        squareMatMulKer<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    #endif

    hipMemcpyAsync( C, Cd, bytesC, hipMemcpyDeviceToHost, strm);

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    free(A);
    free(B);

    return ms;
}


float squareMatMulKer(
    float *Ad, float *Bd, float *Cd, 
    int n,
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{
    float ms;    

    randomMatrix(n,n, Ad);
    randomMatrix(n,n, Bd);  
  
    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 


    checkCuda( hipEventRecord(start,0) );


    #ifdef LOWPAR
        squareMatMulGridStrideKer<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    #else
        squareMatMulKer<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    return ms;
}






float smallMatMulKer(
    float *Ad, float *Bd, float *Cd, float *C,
    int m, int k, int n, 
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{

    float ms;
    int bytesA=m*k*sizeof(float);
    int bytesB=k*n*sizeof(float);
    int bytesC=m*n*sizeof(float);
    float  *A=(float*)calloc(1,bytesA);//new float[M*K];
    float *B=(float*)calloc(1,bytesB);//new float[K*N] ;

    randomMatrix(m,k, A);
    randomMatrix(k,n, B);     

    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    checkCuda( hipEventRecord(start,0) );

    hipMemcpyAsync(Ad, A, bytesA, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB, hipMemcpyHostToDevice, strm);   

    #ifdef LOWPAR
        matMulGridStride<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #else
        matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #endif

    hipMemcpyAsync( C, Cd, bytesC, hipMemcpyDeviceToHost, strm);

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    free(A);
    free(B);

    return ms;
}


float matMulKer(
    float *Ad, float *Bd, float *Cd, 
    int m, int k, int n, 
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{
    float ms;    

    randomMatrix(m,k, Ad);
    randomMatrix(k,n, Bd);     
    
    /*#ifdef LOWPAR
        dim3 dimBlock(4,4,1);
        dim3 dimGrid(1,1,1); 
    #else*/
        dim3 dimBlock(BLOCK,BLOCK,1);
        dim3 dimGrid(GRIDx, GRIDy,1); 
   // #endif

    checkCuda( hipEventRecord(start,0) );


    #ifdef LOWPAR
        matMulGridStride<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #else
        matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    return ms;
}








/*float smallMatMulKer(
    float *Ad, float *Bd, float *Cd, float *C,
    int m, int k, int n, 
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{

    float ms;
    int bytesA=m*k*sizeof(float);
    int bytesB=k*n*sizeof(float);
    int bytesC=m*n*sizeof(float);
    float  *A=(float*)calloc(1,bytesA);//new float[M*K];
    float *B=(float*)calloc(1,bytesB);//new float[K*N] ;

    randomMatrix(m,k, A);
    randomMatrix(k,n, B);     

    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    checkCuda( hipEventRecord(start,0) );

    hipMemcpyAsync(Ad, A, bytesA, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB, hipMemcpyHostToDevice, strm);   

    #ifdef LOWPAR
        matMulGridStride<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #else
        matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #endif

    hipMemcpyAsync( C, Cd, bytesC, hipMemcpyDeviceToHost, strm);

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    free(A);
    free(B);

    return ms;
}*/


float newMatMulKer(
    float *Ad, float *Bd, float *Cd, float *C,
    int m, int k, int n, int chunk,
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{

    float ms;
    int bytesA=m*k*sizeof(float);
    int bytesB=k*n*sizeof(float);
    int bytesC=m*n*sizeof(float);
    float  *A=(float*)calloc(chunk,bytesA);//new float[M*K];
    float *B=(float*)calloc(chunk,bytesB);//new float[K*N] ;

    for(int i=0; i<chunk; ++i){
        randomMatrix(m, k, &A[i*m*k]);
        randomMatrix(k, n, &B[i*k*n]);  
    }
   

    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    checkCuda( hipEventRecord(start,0) );

    hipMemcpyAsync(Ad, A, bytesA, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB, hipMemcpyHostToDevice, strm);   

    #ifdef LOWPAR
        matMulGridStride<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #else
        matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    #endif

    hipMemcpyAsync( C, Cd, bytesC, hipMemcpyDeviceToHost, strm);

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    free(A);
    free(B);

    return ms;
}




float newSquareMatMulKer(
    float *Ad, float *Bd, float *Cd, float *C,
    int n, int chunk,
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{

    float ms;
    int size=n*n;
    int bytesMat=size*sizeof(float);
    //int bytesB=n*n*sizeof(float);
    //int bytesC=n*n*sizeof(float);
    float  *A=(float*)calloc(chunk, bytesMat);//new float[M*K];
    float *B=(float*)calloc(chunk, bytesMat);//new float[K*N] ;

    for(int i=0; i<chunk; ++i){
        randomMatrix(n, n, &A[i*size]);
        randomMatrix(n, n, &B[i*size]);
    }


    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    checkCuda( hipEventRecord(start,0) );

    hipMemcpyAsync(Ad, A, bytesMat, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesMat, hipMemcpyHostToDevice, strm);   

    #ifdef LOWPAR
        squareMatMulGridStrideKer<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    #else
        squareMatMulKer<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    #endif

    hipMemcpyAsync( C, Cd, bytesMat, hipMemcpyDeviceToHost, strm);

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );

    free(A);
    free(B);

    return ms;
}









float blurBoxFilter (
    unsigned char *img_in, unsigned char *img_out,
    int width, int height,
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{    
    float ms=0;
    int bytes=width*height*3*sizeof(unsigned char);
       
    #ifdef LOWPAR
        dim3 blockDims(2,1,1);
        dim3 gridDims(1,1,1); 
    #else
        dim3 blockDims(BLOCK,1,1);
        dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );
    #endif
    checkCuda( hipEventRecord(start,0) ); 

    #ifdef LOWPAR
        blurBoxGridStride<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, width, height); 
    #else
        blurBoxFilterKer<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, width, height); 
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
 
    return ms;
}


float blurGaussianfilter (
    unsigned char *img_in, unsigned char *img_out,
    int width, int height,int kerdim, float sigma,
    hipStream_t strm,
    hipEvent_t start, hipEvent_t stop)
{    

    float *ker;
    int bytes=kerdim*kerdim*sizeof(unsigned char);
    float ms=0;

    checkCuda(hipMallocManaged(&ker, bytes));
    ker=getGaussian(kerdim, sigma);        
       
    #ifdef LOWPAR
        dim3 blockDims(2,2,1);
        dim3 gridDims(1,1, 1 );
    #else
        dim3 blockDims(BLOCK,BLOCK,1);        
        dim3 gridDims((width*3)/blockDims.x, (height*3)/blockDims.y, 1 ); //dim3 gridDims((unsigned int) ceil((double)(bytes/blockDims.x)), 1, 1 );
    #endif
    checkCuda( hipEventRecord(start,0) ); 

    #ifdef LOWPAR
        gaussianBlurGridStride<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, height, width, ker, kerdim);
    #else
        gaussianBlurKer<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, height, width, ker, kerdim);
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
 
    return ms;
}