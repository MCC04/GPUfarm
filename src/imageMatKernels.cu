#include "hip/hip_runtime.h"
#include <math.h>
#include <algorithm>
#include <imageMatrix.h>


#define HIGH 500.0f
#define LOW -500.0f

/*************
 * UTILITIES *
 * ********* */
template<typename T> inline T getMatrixVal(T *mat, int row, int col, int width)
{ return mat[row*width+col]; }

template<typename T> inline void setMatrixVal(T *mat, int row, int col, int width, T val)
{ mat[row*width+col] = val; }

void randomMatrix(const int m, int n,float *mat){
    for(int r=0; r<m; ++r)
        for(int c=0; c<n; ++c){
            int rnd = (float)std::rand();
            float val = LOW + (rnd*(HIGH-LOW)/RAND_MAX);
            setMatrixVal(mat, r, c, n, val);
        }     
}

void getGaussian(float* ker,int dim, float sigma)
{
    float sum=0.0;
    int i,j;
    float sigma2 = sigma*sigma;
    for (i=0 ; i<dim ; i++) {
        for (j=0 ; j<dim ; j++) {
            int i2 = i*i;
            int j2 = j*i;
            int r = std::sqrt(i2+j2);
            //float val = exp((float)(-(i2+j2))/(2*sigma2))/(2*M_PI*sigma2);
            float val = exp((float)(-(r*r))/(2*sigma2))/(2*M_PI*sigma2);

            setMatrixVal<float>(ker,i,j,dim, val);
            sum += val;
        }
    }

    for (i=0 ; i<dim ; i++) 
        for (j=0 ; j<dim ; j++) {
            float val=getMatrixVal<float>(ker,i,j,dim)/sum;
            setMatrixVal<float>(ker,i,j,dim, val);
        }
}


/**********
* KERNELS *
***********/

/**** MATMUL ****/
__global__ void matMulKernel(float* A, float* B, float* C, int m, int k, int n, int chunk) {   
    int ROW = blockIdx.x*blockDim.x+threadIdx.x;
    int COL = blockIdx.y*blockDim.y+threadIdx.y;
 
    if (ROW<m && COL<n) {
        int sizeA = m*k;
        int sizeB = k*n;
        int sizeC = m*n;
        float tmpSum = 0.0f;
        int offsA = 0, offsB = 0, offsC = 0;
        
        for (int s=0; s<chunk; ++s)
        {
            offsA = s*sizeA;
            offsB = s*sizeB;
            offsC = s*sizeC;
            tmpSum = 0.0f;
        
            for (int i = 0; i < k; ++i) {
                tmpSum += A[offsA+(ROW*k)+i] * B[offsB+(i*n)+COL];
            }        
            C[offsC+(ROW*n)+COL] = tmpSum;
        }
    }
    return ;
}

/**** GRID-STRIDE MATMUL ****/
__global__ void matMulGridStride(float* A, float* B, float* C, int m, int k, int n, int chunk) {
    int ROW = blockIdx.x*blockDim.x+threadIdx.x;
    int COL = blockIdx.y*blockDim.y+threadIdx.y;

    int Rstride = blockDim.x*gridDim.x;
    int Cstride = blockDim.y*gridDim.y;    

    float tmpSum = 0.0f;
    int offsA = 0, offsB = 0, offsC = 0;
    for (int s=0; s<chunk; ++s)
    {
        offsA = s*m*k;
        offsB = s*k*n;
        offsC = s*m*n;

        for (int r=ROW; r<m; r+=Rstride) {
            for (int c=COL; c<n; c+=Cstride) {        
               tmpSum=0;
                for (int i = 0; i <k; ++i) {
                    tmpSum += A[offsA+(r*k)+i] * B[offsB+(i*n)+c];
                }
                C[offsC+(r*n)+c] = tmpSum;
            }           
        }
    }    
    return ;
}

/**** SQUARE MATMUL ****/
__global__ void squareMatMulKernel(float* A, float* B, float* C, int N, int chunk) {

    int ROW = blockIdx.x*blockDim.x+threadIdx.x;
    int COL = blockIdx.y*blockDim.y+threadIdx.y;
 
    if (ROW<N && COL<N) {
        int size=N*N;
        float tmpSum=0.0f;        
        for (int s=0; s<chunk; ++s)
        {
            int offs = s*size;
            tmpSum = 0.0f;        
            for (int i = 0; i < N; ++i) {
                tmpSum += A[offs+(ROW*N)+i] * B[offs+(i*N)+COL];
            }        
            C[offs+(ROW*N)+COL] = tmpSum;
        }
    }
    return ;
}

/**** GRID-STRIDE SQUARE MATMUL ****/
__global__ void squareMatMulGridStrideKer(float* A, float* B, float* C, int N, int chunk) {

    int ROW = blockIdx.x*blockDim.x+threadIdx.x;
    int COL = blockIdx.y*blockDim.y+threadIdx.y;

    int Rstride = blockDim.x*gridDim.x;
    int Cstride = blockDim.y*gridDim.y;    

    float tmpSum = 0.0f;
    int offs = 0;
    for (int s=0; s<chunk; ++s)
    {
        offs = s*N*N;
        for (int k=ROW; k<N; k+=Rstride) {
            for (int j=COL; j<N; j+=Cstride) {        
               tmpSum=0;
                for (int i = 0; i < N; i++) {
                    tmpSum += A[offs+(k*N)+i] * B[offs+(i*N)+j];
                }
                C[offs+(k*N)+j] = tmpSum;
            }               
        }
    }    
    return ;
}

/**** BLURBOX ****/
__global__ void blurBoxFilterKer(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x+threadIdx.x;
    int dim = width*height*3;
    if(offset<dim){
        int x = offset % width;
        int y = (offset-x)/width;
        int fsize = 5; // Filter size
        if(offset < width*height) {
            float output_red = 0;
            float output_green = 0;
            float output_blue = 0;
            int hits = 0;
            for(int ox = -fsize; ox < fsize+1; ++ox) {
                for(int oy = -fsize; oy < fsize+1; ++oy) {
                    if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                        const int currentoffset = (offset+ox+oy*width)*3;
                        output_red += input_image[currentoffset]; 
                        output_green += input_image[currentoffset+1];
                        output_blue += input_image[currentoffset+2];
                        hits++;
                    }
                }
            }
            output_image[offset*3] = output_red/hits;
            output_image[offset*3+1] = output_green/hits;
            output_image[offset*3+2] = output_blue/hits;
        }
    }
    return;
}

/**** GRID-STRIDE BLUR BOX ****/
__global__ void blurBoxGridStride(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x;
    int fsize = 5; // Filter size

    for(int i=offset; i<width*height; i+=stride)
    {    
        int x = offset % width;
        int y = (offset-x)/width;

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = ( i +ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[i *3] = output_red/hits;
        output_image[i *3+1] = output_green/hits;
        output_image[i *3+2] = output_blue/hits;
    }
    return ;
}


/**** GAUSSIAN BLUR ****/
__global__ void gaussianBlurKer (const unsigned char* const inputChannel, unsigned char* outputChannel,
                                int numRows, int numCols, const float* filter, const int filterWidth)
{
    /*
     int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    //if (x >= numCols || y >= numRows)
    //    return;
    if (x < numRows && y < numCols){
        int idx = x*numCols+y;
        float blur = 0.0f;
        for (int i = 0; i < filterWidth; ++i) {
            for (int j = 0; j < filterWidth; ++j) {
                int p_x = x+i-filterWidth/2;
                int p_y = y+j-filterWidth/2;
                p_x = min(max(p_x,0), numRows-1);
                p_y = min(max(p_y,0), numCols-1);
                float filter_value = filter[ i*filterWidth+j ];
                blur += filter_value*static_cast<float>(inputChannel[ p_x*numCols+p_y ]);
            }
        }
        outputChannel[idx] = blur;
    }
    return ;
    */



    int r = blockDim.x * blockIdx.x + threadIdx.x;
    int c = blockDim.y * blockIdx.y + threadIdx.y;
    //if (x >= numCols || y >= numRows)
    //    return;
    if (r < numCols && c < numRows){
        int idx = c*numCols+r;
        float blur = 0.0f;
        for (int fr = 0; fr < filterWidth; ++fr) {
            for (int fc = 0; fc < filterWidth; ++fc) {
                int p_x = r+fr-filterWidth/2;
                int p_y = c+fc-filterWidth/2;
                p_x = min(max(p_x,0), numCols-1);
                p_y = min(max(p_y,0), numRows-1);
                float filter_value = filter[ fc*filterWidth+fr ];
                blur += filter_value*static_cast<float>(inputChannel[ p_y*numCols+p_x ]);
            }
        }
       // outputChannel[idx] = static_cast<unsigned short>(blur);
       outputChannel[idx] = blur;
    }
    return ;
}

/**** GRID-STRIDE GAUSSIAN BLUR ****/
__global__ void gaussianBlurGridStride(
 const unsigned char* const inputChannel,
unsigned char* outputChannel,
int numRows, int numCols,
const float* filter, const int filterWidth)
{
    const unsigned int indexX = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int indexY = blockDim.y * blockIdx.y + threadIdx.y;

    const unsigned int strideX = blockDim.x * gridDim.x;
    const unsigned int strideY = blockDim.y * gridDim.y;

    for(int k=indexX; k<numCols; k+=strideX)
    {
        for(int l=indexY; l<numRows; l+=strideY)
        {
            int idx = l * numCols + k;
            float blur = 0.0f;
            for (int i = 0; i < filterWidth; i++) {
                for (int j = 0; j < filterWidth; j++) {
                    int p_x = k + i - filterWidth/2;
                    int p_y = l + j - filterWidth/2;
                    p_x = min(max(p_x, 0), numCols - 1);
                    p_y = min(max(p_y, 0), numRows - 1);
                    float filter_value = filter[i * filterWidth + j];
                    blur += filter_value *
                    static_cast<float>(inputChannel[p_y * numCols + p_x]);
                }
            }
            outputChannel[idx] = blur;
        }
    }
    return ;
}


/*******************
**KERNEL LAUNCHERS**
********************/

/**** NON SQUARE MATMUL ****/
#ifdef MATMUL
void newMatMulKer(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, 
        int m, int k, int n, int chunk, hipStream_t strm)
{
    int bytesA = m*k*sizeof(float);
    int bytesB = k*n*sizeof(float);
    int bytesC = m*n*sizeof(float);

    hipMemcpyAsync(Ad, A, bytesA*chunk, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB*chunk, hipMemcpyHostToDevice, strm);   

    dim3 dimBlock( BLOCK,BLOCK,1 );
    #ifdef LOWPAR
        GRIDx = 1;
        GRIDy = 1;
        dim3 dimGrid( GRIDx,GRIDy,1 ); 
        matMulGridStride<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n, chunk);
    #else
        int sizeX,sizeY;
        if (m%BLOCK == 0) sizeX = m;
        else sizeX = m+BLOCK-1;

        if (n%BLOCK == 0) sizeY = n;
        else sizeY = n+BLOCK-1;

        GRIDx = (sizeX)/BLOCK;
        GRIDy = (sizeY)/BLOCK;
        dim3 dimGrid( GRIDx,GRIDy,1 ); 
        matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n, chunk);
    #endif

    hipMemcpyAsync( C, Cd, bytesC*chunk, hipMemcpyDeviceToHost, strm);

    //hipDeviceSynchronize();
}

/**** SQUARE MATMUL ****/
void newSquareMatMulKer(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, 
            int n, int chunk, hipStream_t strm)
{
    int size = n*n;
    int bytesMat = size*sizeof(float);

    gpuErrchk( hipMemcpyAsync(Ad, A, bytesMat*chunk, hipMemcpyHostToDevice, strm) );    
    gpuErrchk( hipMemcpyAsync(Bd, B, bytesMat*chunk, hipMemcpyHostToDevice, strm) );   

    dim3 dimBlock( BLOCK,BLOCK,1 );
    #ifdef LOWPAR        
        GRIDx = 1;
        dim3 dimGrid( GRIDx,GRIDx,1 ); 
        squareMatMulGridStrideKer<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n, chunk);
    #else
        int sizeX;
        if (n%BLOCK == 0) sizeX = n;
        else sizeX = n+BLOCK-1;

        GRIDx = (sizeX)/BLOCK;
        dim3 dimGrid( GRIDx,GRIDx,1 ); 
        squareMatMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n, chunk);
    #endif
    squareMatMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n, chunk);

    gpuErrchk( hipMemcpyAsync( C, Cd, bytesMat*chunk, hipMemcpyDeviceToHost, strm) );

    //hipDeviceSynchronize();
}
#endif

/**** BLURBOX ****/
#ifdef BLURBOX
void blurBoxFilter (unsigned char *img_in, unsigned char *img_out, unsigned char *in_d, unsigned char *out_d, 
                    int width, int height, int bytesSize, hipStream_t strm)
{          
    dim3 blockDims( BLOCK,1,1 );
    gpuErrchk( hipMemcpyAsync(in_d,img_in, bytesSize, hipMemcpyHostToDevice, strm) );  
    #ifdef LOWPAR   
        GRIDx = 1;     
        dim3 gridDims( GRIDx,1,1 ); 
        blurBoxGridStride<<<gridDims, blockDims, 0, strm>>>(in_d, out_d, width, height); 
    #else
        GRIDx = (unsigned int)((bytesSize+BLOCK-1)/blockDims.x);
        dim3 gridDims( GRIDx,1,1 );
        blurBoxFilterKer<<<gridDims, blockDims, 0, strm>>>(in_d, out_d, width, height); 
    #endif
    gpuErrchk( hipMemcpyAsync( img_out, out_d, bytesSize, hipMemcpyDeviceToHost, strm) );
    hipDeviceSynchronize();

}
#endif

/**** BLURGAUSS ****/
#ifdef BLURGAUSS
void blurGaussianfilter (unsigned char *img_in, unsigned char *img_out, unsigned char *in_d, unsigned char *out_d, 
                        float *ker_d, int width, int height, int bytesSize, int kerdim, hipStream_t strm)
{    
    dim3 blockDims( BLOCK,BLOCK,1 );
    gpuErrchk( hipMemcpyAsync(in_d, img_in, bytesSize, hipMemcpyHostToDevice, strm) );    
    #ifdef LOWPAR        
        dim3 gridDims( 1,1,1 );
        gaussianBlurGridStride<<<gridDims, blockDims, 0, strm>>>(in_d, out_d, height, width, ker_d, kerdim);
   #else
        dim3 gridDims(((width*3)+blockDims.x-1)/blockDims.x, ((height*3)+blockDims.y-1)/blockDims.y, 1 ); 
        gaussianBlurKer<<<gridDims, blockDims, 0, strm>>>(in_d, out_d, height, width, ker_d, kerdim);
    #endif
    gpuErrchk( hipMemcpyAsync( img_out, out_d, bytesSize, hipMemcpyDeviceToHost, strm) );
    hipDeviceSynchronize();
}
#endif