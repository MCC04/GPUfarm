#include "hip/hip_runtime.h"
#include <math.h>
#include <algorithm>
#include <imageMatrix.h>


#define HIGH 500.0f
#define LOW -500.0f

/* ********* *
 * UTILITIES *
 * ********* */
template<typename T> inline T getMatrixVal(T *mat, int row, int col, int width)
{ return mat[row*width+col]; }

template<typename T> inline void setMatrixVal(T *mat, int row, int col, int width, T val)
{ mat[row*width+col] = val; }

void randomMatrix(const int m, int n,float *mat){
    for(int r=0; r<m; ++r)
        for(int c=0; c<n; ++c){
            int rnd = (float)std::rand();
            float val = LOW + (rnd*(HIGH-LOW)/RAND_MAX);
            setMatrixVal(mat, r, c, n, val);
        }     
}

void launchConfig(int m, int n){
    #ifdef LOWPAR
        GRIDx = 1;
        GRIDy = 1;
    #else
        int sizeX,sizeY;
        if (m%BLOCK == 0) sizeX = m;
        else sizeX = m+BLOCK-1;

        if (n%BLOCK == 0) sizeY = n;
        else sizeY = n+BLOCK-1;

        GRIDx = (sizeX)/BLOCK;
        GRIDy = (sizeY)/BLOCK;
    #endif
}

void getGaussian(float* ker,int dim, float sigma)
{
    float sum=0.0;
    int i,j;
    float sigma2 = sigma*sigma;
    for (i=0 ; i<dim ; i++) {
        for (j=0 ; j<dim ; j++) {
            int i2 = i*i;
            int j2 = j*i;
            int r = std::sqrt(i2+j2);
            float val = exp((float)(-(r*r))/(2*sigma2))/(2*M_PI*sigma2);

            setMatrixVal<float>(ker,i,j,dim, val);
            sum += val;
        }
    }

    for (i=0 ; i<dim ; i++) 
        for (j=0 ; j<dim ; j++) {
            float val=getMatrixVal<float>(ker,i,j,dim)/sum;
            setMatrixVal<float>(ker,i,j,dim, val);
        }
}


/* ******* *
 * KERNELS *
 * ******* */

/**** MATMUL ****/
__global__ void matMulKernel(float* A, float* B, float* C, int m, int k, int n) {   
    int ROW = blockIdx.x*blockDim.x+threadIdx.x;
    int COL = blockIdx.y*blockDim.y+threadIdx.y;
 
    if (ROW<m && COL<n) {
        float tmpSum = 0.0f;        
        for (int i = 0; i < k; ++i) {
            tmpSum += A[(ROW*k)+i] * B[(i*n)+COL];
        }        
        C[(ROW*n)+COL] = tmpSum;
    }
    return ;
}

/**** SQUARE MATMUL ****/
__global__ void squareMatMulKernel(float* A, float* B, float* C, int N) {
    int COL = blockIdx.x*blockDim.x+threadIdx.x;
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
 
    if (ROW<N && COL<N) {
        float tmpSum=0.0f;        

        for (int i = 0; i < N; ++i) {
            tmpSum += A[(ROW*N)+i] * B[(i*N)+COL];
        }        
        C[(ROW*N)+COL] = tmpSum;        
    }
    return ;
}

#define TILE_WIDTH 16
/***** SHARED MATMUL *****/
__global__ void sharedMatMulKernel(float *A, float *B, float *C, int size)
{
    //const int blockSize = blockDim.x;
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = size * TILE_WIDTH * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + size - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = TILE_WIDTH;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = TILE_WIDTH * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = TILE_WIDTH * size;

    // The element of the block sub-matrix that is computed by the thread
    float Csub = 0;



    // Loop over all the sub-matrices of A and B required to compute the block sub-matrix

    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
    {
        // Shared memory for the sub-matrices of A and B
        __shared__ float As[TILE_WIDTH][TILE_WIDTH];
        __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];


        // Load the matrices from global memory to shared memory, each thread loads one element of each matrix
        As[ty][tx] = A[a + size * ty + tx];
        Bs[ty][tx] = B[b + size * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        #pragma unroll
        // Multiply the two matrices together, each thread computes one element of the block sub-matrix
        for (int k = 0; k < TILE_WIDTH; ++k) 
            Csub += As[ty][k] * Bs[k][tx];


        // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to global memory, each thread writes one element
    int c = size * TILE_WIDTH * by + TILE_WIDTH * bx;
    C[c + size * ty + tx] = Csub;
}







/**** GRID-STRIDE SQUARE MATMUL ****/
/* __global__ void squareMatMulGridStrideKer(float* A, float* B, float* C, int N) {

    int ROW = blockIdx.x*blockDim.x+threadIdx.x;
    int COL = blockIdx.y*blockDim.y+threadIdx.y;

    int Rstride = blockDim.x*gridDim.x;
    int Cstride = blockDim.y*gridDim.y;    

    float tmpSum = 0.0f;

    for (int k=ROW; k<N; k+=Rstride) {
        for (int j=COL; j<N; j+=Cstride) {        
            tmpSum=0;
            for (int i = 0; i < N; i++) {
                tmpSum += A[(k*N)+i] * B[(i*N)+j];
            }
            C[(k*N)+j] = tmpSum;
        }               
    }
  
    return ;















 //const int blockSize = blockDim.x;
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = size * TILE_WIDTH * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + size - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = TILE_WIDTH;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = TILE_WIDTH * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = TILE_WIDTH * size;

    // The element of the block sub-matrix that is computed by the thread
    float Csub = 0;



    // Loop over all the sub-matrices of A and B required to compute the block sub-matrix

    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
    {
        // Shared memory for the sub-matrices of A and B
        __shared__ float As[TILE_WIDTH][TILE_WIDTH];
        __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];


        // Load the matrices from global memory to shared memory, each thread loads one element of each matrix
        As[ty][tx] = A[a + size * ty + tx];
        Bs[ty][tx] = B[b + size * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        #pragma unroll
        // Multiply the two matrices together, each thread computes one element of the block sub-matrix
        for (int k = 0; k < TILE_WIDTH; ++k) 
            Csub += As[ty][k] * Bs[k][tx];


        // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to global memory, each thread writes one element
    int c = size * TILE_WIDTH * by + TILE_WIDTH * bx;
    C[c + size * ty + tx] = Csub;












}*/


/**** BLURBOX ****/
__global__ void blurBoxFilterKer(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x+threadIdx.x;
    int dim = width*height*3;
    if(offset<dim){
        int x = offset % width;
        int y = (offset-x)/width;
        int fsize = 5; // Filter size
        if(offset < width*height) {
            float output_red = 0;
            float output_green = 0;
            float output_blue = 0;
            int hits = 0;
            for(int ox = -fsize; ox < fsize+1; ++ox) {
                for(int oy = -fsize; oy < fsize+1; ++oy) {
                    if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                        const int currentoffset = (offset+ox+oy*width)*3;
                        output_red += input_image[currentoffset]; 
                        output_green += input_image[currentoffset+1];
                        output_blue += input_image[currentoffset+2];
                        hits++;
                    }
                }
            }
            output_image[offset*3] = output_red/hits;
            output_image[offset*3+1] = output_green/hits;
            output_image[offset*3+2] = output_blue/hits;
        }
    }
    return;
}


/**** GRID-STRIDE MATMUL ****/
__global__ void matMulGridStride(float* A, float* B, float* C, int m, int k, int n) {
    int ROW = blockIdx.x*blockDim.x+threadIdx.x;
    int COL = blockIdx.y*blockDim.y+threadIdx.y;

    int Rstride = blockDim.x*gridDim.x;
    int Cstride = blockDim.y*gridDim.y;    

    for (int r=ROW; r<m; r+=Rstride) {
        for (int c=COL; c<n; c+=Cstride) {        
            float tmpSum = 0.0f;
            for (int i = 0; i <k; ++i) {
                tmpSum += A[(r*k)+i] * B[(i*n)+c];
            }
            C[(r*n)+c] = tmpSum;
        }           
    }   
    return ;
}

/**** GRID-STRIDE SQUARE MATMUL ****/
__global__ void squareMatMulGridStrideKer(float* A, float* B, float* C, int N) {

    int ROW = blockIdx.x*blockDim.x+threadIdx.x;
    int COL = blockIdx.y*blockDim.y+threadIdx.y;

    int Rstride = blockDim.x*gridDim.x;
    int Cstride = blockDim.y*gridDim.y;    

    float tmpSum = 0.0f;
    for (int k=ROW; k<N; k+=Rstride) {
        for (int j=COL; j<N; j+=Cstride) {        
            tmpSum=0;
            for (int i = 0; i < N; i++) {
                tmpSum += A[(k*N)+i] * B[(i*N)+j];
            }
            C[(k*N)+j] = tmpSum;
        }               
    }  
    return ;
}

/**** GRID-STRIDE BLUR BOX ****/
__global__ void blurBoxGridStride(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x;
    int fsize = 5; // Filter size

    for(int i=offset; i<width*height; i+=stride)
    {    
        int x = offset % width;
        int y = (offset-x)/width;

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = ( i +ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[i *3] = output_red/hits;
        output_image[i *3+1] = output_green/hits;
        output_image[i *3+2] = output_blue/hits;
    }
    return ;
}


/*******************
**KERNEL LAUNCHERS**
********************/

/**** SQUARE MATMUL ****/
void streamSquareMatMul(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, 
            int n, hipStream_t strm, bool shared)
{
    int size = n*n;
    int bytesMat = size*sizeof(float);
    // H2D memCopy
    gpuErrchk( hipMemcpyAsync(Ad, A, bytesMat, hipMemcpyHostToDevice, strm) );    
    gpuErrchk( hipMemcpyAsync(Bd, B, bytesMat, hipMemcpyHostToDevice, strm) );   
    // Grid and Block setting
    launchConfig(n, n);
    dim3 dimBlock( BLOCK,BLOCK,1 );
    /*#ifdef LOWPAR        
        GRIDx = 1;
    #else
        int sizeX;
        if (n%BLOCK == 0) sizeX = n;
        else sizeX = n+BLOCK-1;
        GRIDx = sizeX/BLOCK;
    #endif*/
    dim3 dimGrid( GRIDx,GRIDx,1 ); 
    // Kernel launch
    if(shared)
        sharedMatMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    else
        squareMatMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    // D2H memCopy
    gpuErrchk( hipMemcpyAsync( C, Cd, bytesMat, hipMemcpyDeviceToHost, strm) );

    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
    #endif 
}

void squareMatMul(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, int n, bool shared)
{
    int size = n*n;
    int bytesMat = size*sizeof(float);

    gpuErrchk( hipMemcpy(Ad, A, bytesMat, hipMemcpyHostToDevice) );    
    gpuErrchk( hipMemcpy(Bd, B, bytesMat, hipMemcpyHostToDevice) );   

    launchConfig(n, n);
    dim3 dimBlock( BLOCK,BLOCK,1 );
    /*#ifdef LOWPAR        
        GRIDx = 1;
    #else
        int sizeX;
        if (n%BLOCK == 0) sizeX = n;
        else sizeX = n+BLOCK-1;

        GRIDx = (sizeX)/BLOCK;
    #endif*/
    dim3 dimGrid( GRIDx,GRIDx,1 ); 
    if(shared)
        sharedMatMulKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, n);
    else
        squareMatMulKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, n);
    //squareMatMulGridStrideKer<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, n);

    gpuErrchk( hipMemcpy( C, Cd, bytesMat, hipMemcpyDeviceToHost) );

    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
    #endif 
}





/**** NON SQUARE MATMUL ****/
#ifdef MATMUL
void streamMatMul(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, 
        int m, int k, int n, hipStream_t strm)
{
    int bytesA = m*k*sizeof(float);
    int bytesB = k*n*sizeof(float);
    int bytesC = m*n*sizeof(float);
    // H2D memCopy
    hipMemcpyAsync(Ad, A, bytesA, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB, hipMemcpyHostToDevice, strm);   
    // Grid and Block setting    
    launchConfig(m, n);
    dim3 dimBlock( BLOCK,BLOCK,1 );
    /*#ifdef LOWPAR
        GRIDx = 1;
        GRIDy = 1;
    #else
        int sizeX,sizeY;
        if (m%BLOCK == 0) sizeX = m;
        else sizeX = m+BLOCK-1;

        if (n%BLOCK == 0) sizeY = n;
        else sizeY = n+BLOCK-1;

        GRIDx = (sizeX)/BLOCK;
        GRIDy = (sizeY)/BLOCK;
    #endif*/
    dim3 dimGrid( GRIDx,GRIDy,1 ); 
    // Kernel launch
    matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    // D2H memCopy
    hipMemcpyAsync( C, Cd, bytesC, hipMemcpyDeviceToHost, strm);
}


void matMul(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, 
        int m, int k, int n)
{
    int bytesA = m*k*sizeof(float);
    int bytesB = k*n*sizeof(float);
    int bytesC = m*n*sizeof(float);
    // H2D memCopy
    hipMemcpy(Ad, A, bytesA, hipMemcpyHostToDevice);    
    hipMemcpy(Bd, B, bytesB, hipMemcpyHostToDevice);   
    // Grid and Block setting
    launchConfig(m, n);
    dim3 dimBlock( BLOCK,BLOCK,1 );
    /*#ifdef LOWPAR
        GRIDx = 1;
        GRIDy = 1;
    #else
        int sizeX,sizeY;
        if (m%BLOCK == 0) sizeX = m;
        else sizeX = m+BLOCK-1;

        if (n%BLOCK == 0) sizeY = n;
        else sizeY = n+BLOCK-1;

        GRIDx = (sizeX)/BLOCK;
        GRIDy = (sizeY)/BLOCK;
    #endif*/
    dim3 dimGrid( GRIDx,GRIDy,1 ); 
    // Kernel launch
    matMulKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, m,  k,  n);
    // D2H memCopy
    hipMemcpy( C, Cd, bytesC, hipMemcpyDeviceToHost);
}



/* SHARED MEMORY MATMUL 
void squareSharedMatMul(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, int n)
{
    int size = n*n;
    int bytesMat = size*sizeof(float);

    gpuErrchk( hipMemcpy(Ad, A, bytesMat, hipMemcpyHostToDevice) );    
    gpuErrchk( hipMemcpy(Bd, B, bytesMat, hipMemcpyHostToDevice) );   

    launchConfig(n, n);
    dim3 dimBlock( BLOCK,BLOCK,1 );
    /*#ifdef LOWPAR        
        GRIDx = 1;
    #else
        int sizeX;
        if (n%BLOCK == 0) sizeX = n;
        else sizeX = n+BLOCK-1;

        GRIDx = (sizeX)/BLOCK;
    #endif
    dim3 dimGrid( GRIDx,GRIDx,1 ); 
    sharedMatMulKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, n);

    gpuErrchk( hipMemcpy( C, Cd, bytesMat, hipMemcpyDeviceToHost) );
}*/


/* void streamSquareSharedMatMul(float *A, float B*, float C*, float *Ad, float *Bd, float *Cd, int n, hipStream_t strm){
    int size = n*n;
    int bytesMat = size*sizeof(float);
    // H2D memCopy
    gpuErrchk( hipMemcpyAsync(Ad, A, bytesMat, hipMemcpyHostToDevice, strm) );    
    gpuErrchk( hipMemcpyAsync(Bd, B, bytesMat, hipMemcpyHostToDevice, strm) );   
    // Grid and Block setting
    launchConfig(n, n);
    dim3 dimBlock( BLOCK,BLOCK,1 );
    /*#ifdef LOWPAR        
        GRIDx = 1;
    #else
        int sizeX;
        if (n%BLOCK == 0) sizeX = n;
        else sizeX = n+BLOCK-1;
        GRIDx = sizeX/BLOCK;
    #endif
    dim3 dimGrid( GRIDx,GRIDx,1 ); 
    // Kernel launch
    sharedMatMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    // D2H memCopy
    gpuErrchk( hipMemcpyAsync( C, Cd, bytesMat, hipMemcpyDeviceToHost, strm) );
    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
    #endif 
}*/

#endif


/**** BLURBOX ****/
#ifdef BLURBOX
void streamBlurBoxFilter (unsigned char *in_h, unsigned char *out_h, unsigned char *in_d, unsigned char *out_d, 
                    int width, int height, hipStream_t strm)
{          
    int size = width*height*3;
    // H2D memCopy
    gpuErrchk( hipMemcpyAsync(in_d, in_h, size, hipMemcpyHostToDevice, strm) );  
    // Grid and Block setting
    #ifdef LOWPAR   
        GRIDx = 1;     
        //blurBoxGridStride<<<gridDims, blockDims, 0, strm>>>(in_d, out_d, width, height); 
    #else
        GRIDx = (unsigned int)((size+BLOCK-1)/BLOCK);
    #endif
    dim3 blockDims( BLOCK,1,1 );
    dim3 gridDims( GRIDx,1,1 );
    // Kernel launch
    #ifdef LOWPAR   
        blurBoxGridStride<<<gridDims, blockDims, 0, strm>>>(in_d, out_d, width, height); 
    #else
        blurBoxFilterKer<<<gridDims, blockDims, 0, strm>>>(in_d, out_d, width, height); 
    #endif
    //blurBoxFilterKer<<<gridDims, blockDims, 0, strm>>>(in_d, out_d, width, height); 
    // D2H memCopy
    gpuErrchk( hipMemcpyAsync( out_h, out_d, size, hipMemcpyDeviceToHost, strm) );
    //hipDeviceSynchronize();
    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
    #endif 

}

void blurBoxFilter (unsigned char *in_h, unsigned char *out_h, unsigned char *in_d, unsigned char *out_d, 
                    int width, int height)
{     
    int size = width*height*3;    
    // H2D memCopy
    gpuErrchk( hipMemcpy(in_d,in_h, size, hipMemcpyHostToDevice) );   
    // Grid and Block setting    
     
    #ifdef LOWPAR   
        GRIDx = 1;           
        //blurBoxGridStride<<<gridDims, blockDims>>>(in_d, out_d, width, height); 
    #else
        GRIDx = (unsigned int)((size+BLOCK-1)/BLOCK);
        //dim3 gridDims( GRIDx,1,1 );
    #endif
    dim3 blockDims( BLOCK,1,1 );   
    dim3 gridDims( GRIDx,1,1 ); 
    // Kernel launch
    #ifdef LOWPAR   
        blurBoxGridStride<<<gridDims, blockDims>>>(in_d, out_d, width, height); 
    #else
        blurBoxFilterKer<<<gridDims, blockDims>>>(in_d, out_d, width, height); 
    #endif
    //blurBoxFilterKer<<<gridDims, blockDims>>>(in_d, out_d, width, height); 
    // D2H memCopy
    gpuErrchk( hipMemcpy( out_h, out_d, size, hipMemcpyDeviceToHost) );
}
#endif

/**** BLURGAUSS ****/
/*#ifdef BLURGAUSS
void blurGaussianfilter (unsigned char *img_in, unsigned char *img_out, unsigned char *in_d, unsigned char *out_d, 
                        float *ker_d, int width, int height, int bytesSize, int kerdim, hipStream_t strm)
{    
    dim3 blockDims( BLOCK,BLOCK,1 );
    gpuErrchk( hipMemcpyAsync(in_d, img_in, bytesSize, hipMemcpyHostToDevice, strm) );    
    #ifdef LOWPAR        
        dim3 gridDims( 1,1,1 );
        gaussianBlurGridStride<<<gridDims, blockDims, 0, strm>>>(in_d, out_d, height, width, ker_d, kerdim);
   #else
        dim3 gridDims(((width*3)+blockDims.x-1)/blockDims.x, ((height*3)+blockDims.y-1)/blockDims.y, 1 ); 
        gaussianBlurKer<<<gridDims, blockDims, 0, strm>>>(in_d, out_d, height, width, ker_d, kerdim);
    #endif
    gpuErrchk( hipMemcpyAsync( img_out, out_d, bytesSize, hipMemcpyDeviceToHost, strm) );
    hipDeviceSynchronize();
}
#endif*/