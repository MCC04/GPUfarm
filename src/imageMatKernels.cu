#include "hip/hip_runtime.h"
//#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <assert.h> 
#include <cstdlib>
#include <algorithm>
#include <ctime>
#include <vector>
#include <future>
#include <iterator>
#include <cudaUtils.h>
#include <imageMatrix.h>
//#include <cosFutStr.h>

#define HIGH 500.0f
#define LOW -500.0f

/*auto getMatrixVal(auto *mat, int row, int col, int width)
{
    return mat[row + col*width];
}

void setMatrixVal(auto *mat, int row, int col, int width, auto val)
{   
    mat[row + col*width] = val;
}*/

template<typename T> T getMatrixVal(T *mat, int row, int col, int width)
{
    return mat[row*width + col];
}

template<typename T> void setMatrixVal(T *mat, int row, int col, int width, T val)
{   
    mat[row*width + col] = val;
}

void randomMatrix(const int m, int n,float *mat){
    #ifndef MEASURES
        std::cout<< "MATRIX M: "<<std::endl;  
    #endif

    for(int r = 0; r<m; ++r){
        for(int c = 0; c<n; ++c){
            float val=LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;
            setMatrixVal(mat, r, c, n, val);

            #ifndef MEASURES
                std::cout<< getMatrixVal<float>(mat,r,c,n) << ", ";              
            #endif
        }
        #ifndef MEASURES
            std::cout<< std::endl;  
        #endif
    }       
}

float* getGaussian(int dim, float sigma)
{
    //Matrix kernel(height, Array(width));
    float *ker=new float[dim*dim];
    float sum=0.0;
    int i,j;

    for (i=0 ; i<dim ; i++) {
        for (j=0 ; j<dim ; j++) {
            float val = exp((float)(-(i*i+j*j))/(2*sigma*sigma))
                        /(2*M_PI*sigma*sigma);
            setMatrixVal<float>(ker,i,j,dim, val);
            //kernel[i][j] = exp(-(i*i+j*j)/(2*sigma*sigma))/(2*M_PI*sigma*sigma);
            sum += val;
        }
    }

    for (i=0 ; i<dim ; i++) {
        for (j=0 ; j<dim ; j++) {
            float val=getMatrixVal<float>(ker,i,j,dim)/sum;
            setMatrixVal<float>(ker,i,j,dim, val);
            //kernel[i][j] /= sum;
        }
    }

    return ker;
}

/*********
**KERNELS*
**********/
__global__ void matMulKernel(float* Ad, float* Bd, float* Cd, int m, int k, int n)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if(row<m && col<n) {
        float sum = 0;
        for(int j=0;j<k;j++) {
            sum += Ad[row*k+j] * Bd[j*n+col];
        }
        Cd[row*n+col] = sum;
    }
}

__global__ void blurBoxFilterKer(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 5; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
        }
}

__global__ void gaussianBlurKer(
 const unsigned char* const inputChannel,
unsigned char* outputChannel,
int numRows, int numCols,
const float* filter, const int filterWidth)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= numCols || y >= numRows)
        return;
    int idx = y * numCols + x;
    float blur = 0.0f;
    for (int i = 0; i < filterWidth; i++) {
        for (int j = 0; j < filterWidth; j++) {
            int p_x = x + i - filterWidth/2;
            int p_y = y + j - filterWidth/2;
            p_x = min(max(p_x, 0), numCols - 1);
            p_y = min(max(p_y, 0), numRows - 1);
            float filter_value = filter[i * filterWidth + j];
            blur += filter_value *
            static_cast<float>(inputChannel[p_y * numCols + p_x]);
        }
    }
    outputChannel[idx] = blur;
}


//KERNEL LAUNCERS
float matMulKer(
    float *Ad, float *Bd, float *Cd, 
    int m, int k, int n, 
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{

    float ms;
    /*#ifdef LOWPAR
        dim3 dimBlock(4,4,1);
        dim3 dimGrid(1,1,1); 
    #else
        dim3 dimBlock(BLOCK,BLOCK,1);
        dim3 dimGrid((m+dimBlock.x-1)/dimBlock.x, (n+dimBlock.y-1)/dimBlock.y,1); 
    #endif*/

    dim3 dimBlock(BLOCK,BLOCK,1);
    dim3 dimGrid(GRIDx, GRIDy,1); 

    

    randomMatrix(m,k, Ad);
    randomMatrix(k,n, Bd); 

    checkCuda( hipEventRecord(start,0) );

    matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );




/*
    std::cout<<"Product of the two matrices is:"<<std::endl;
    for(int i=0; i<m; ++i){
        for(int j=0; j<n; ++j){
            for(int r=0; r<k; ++r) {
                float a=getMatrixVal(Ad,i,r,k);
                float b = getMatrixVal(Bd,r,j,n);
                setMatrixVal(Cd,i,j,n,a*b);
                //Cd[i][j]+=Ad[i][r]*Bd[r][j];
            }
            std::cout<<getMatrixVal(Cd,i,j,n)<<" ";
        }
        std::cout<< std::endl;
    }
        
       */      

    







    return ms;
}

float blurBoxFilter (
    unsigned char *img_in, unsigned char *img_out,
    int width, int height,
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{    
    /*for(int i = 0; i < 100; ++i) {
           
    //input_image[where] = in.at(i);
       // std::cout<<std::endl<<"kernel img at "<<i<<": "<< (int)img_in[i];

       // ++where;
    }  */

    float ms=0;
    int bytes=width*height*3*sizeof(unsigned char);
       
    #ifdef LOWPAR
        dim3 dimBlock(32,1,1);
        dim3 dimGrid(1,1,1); 
    #else
        dim3 blockDims(BLOCK,1,1);
        dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );
    #endif
    checkCuda( hipEventRecord(start,0) ); 

    //checkCuda(hipMallocManaged(&img_in, bytes));
    //checkCuda(hipMallocManaged(&img_out, bytes));

    blurBoxFilterKer<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, width, height); 

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
 
    return ms;
}


float blurGaussianfilter (
    unsigned char *img_in, unsigned char *img_out,
    int width, int height,int kerdim, float sigma,
    hipStream_t strm,
    hipEvent_t start, hipEvent_t stop)
{    

    float *ker;
    int bytes=kerdim*kerdim*sizeof(unsigned char);

    checkCuda(hipMallocManaged(&ker, bytes));
    ker=getGaussian(kerdim, sigma);
    

    float ms=0;
       
    dim3 blockDims(16,16,1);
    //dim3 gridDims((unsigned int) ceil((double)(bytes/blockDims.x)), 1, 1 );
    dim3 gridDims((width*3)/blockDims.x, (height*3)/blockDims.y, 1 );

    checkCuda( hipEventRecord(start,0) ); 

    /*checkCuda(hipMallocManaged(&img_in, bytes));
    checkCuda(hipMallocManaged(&img_out, bytes));*/
    

    //blurKernel<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, width, height); 

    /*gaussianBlurKer<<<gridDims, blockDims, 0, strm>>>(
    const unsigned char* const inputChannel,
unsigned char* const outputChannel,
int numRows, int numCols,
const float* const filter, const int filterWidth)*/

    gaussianBlurKer<<<gridDims, blockDims, 0, strm>>>(
        img_in, img_out, height, width, ker, kerdim);
   



    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
 
    return ms;
}