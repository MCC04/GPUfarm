#include "hip/hip_runtime.h"
#include <math.h>
#include <algorithm>
#include <imageMatrix.h>


/* ******* *
 * KERNELS *
 * ******* */

/**** MATMUL ****/
__global__ void matMulKernel(float* A, float* B, float* C, int m, int k, int n) {   
    int COL = blockIdx.x*blockDim.x+threadIdx.x;
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
 
    if (ROW<m && COL<n) {
        float tmpSum = 0.0f;        
        for (int i = 0; i < k; ++i) {
            tmpSum += A[(ROW*k)+i] * B[(i*n)+COL];
        }        
        C[(ROW*n)+COL] = tmpSum;
    }
    return ;
}

/**** SQUARE MATMUL ****/
__global__ void squareMatMulKernel(float* A, float* B, float* C, int N) {
    int COL = blockIdx.x*blockDim.x+threadIdx.x;
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
 
    if (ROW<N && COL<N) {
        float tmpSum=0.0f;        

        for (int i = 0; i < N; ++i) {
            tmpSum += A[(ROW*N)+i] * B[(i*N)+COL];
        }        
        C[(ROW*N)+COL] = tmpSum;        
    }
    return ;
}


/***** SHARED MATMUL *****/
#define TILE_WIDTH 16
__global__ void sharedMatMulKernel(float *A, float *B, float *C, int size)
{
    //const int blockSize = blockDim.x;
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = size * TILE_WIDTH * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + size - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = TILE_WIDTH;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = TILE_WIDTH * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = TILE_WIDTH * size;

    // The element of the block sub-matrix that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
    {
        // Shared memory for the sub-matrices of A and B
        __shared__ float As[TILE_WIDTH][TILE_WIDTH];
        __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

        // Load the matrices from global memory to shared memory, each thread loads one element of each matrix
        As[ty][tx] = A[a + size * ty + tx];
        Bs[ty][tx] = B[b + size * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        #pragma unroll
        // Multiply the two matrices together, each thread computes one element of the block sub-matrix
        for (int k = 0; k < TILE_WIDTH; ++k) 
            Csub += As[ty][k] * Bs[k][tx];

        // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    // Write the block sub-matrix to global memory, each thread writes one element
    int c = size * TILE_WIDTH * by + TILE_WIDTH * bx;
    C[c + size * ty + tx] = Csub;
}


/**** BLURBOX ****/
__global__ void blurBoxFilterKer(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x+threadIdx.x;
    int dim = width*height*3;
    if(offset<dim){
        int x = offset % width;
        int y = (offset-x)/width;
        int fsize = 5; // Filter size
        if(offset < width*height) {
            float output_red = 0;
            float output_green = 0;
            float output_blue = 0;
            int hits = 0;
            for(int ox = -fsize; ox < fsize+1; ++ox) {
                for(int oy = -fsize; oy < fsize+1; ++oy) {
                    if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                        const int currentoffset = (offset+ox+oy*width)*3;
                        output_red += input_image[currentoffset]; 
                        output_green += input_image[currentoffset+1];
                        output_blue += input_image[currentoffset+2];
                        hits++;
                    }
                }
            }
            output_image[offset*3] = output_red/hits;
            output_image[offset*3+1] = output_green/hits;
            output_image[offset*3+2] = output_blue/hits;
        }
    }
    return;
}


/**** GRID-STRIDE MATMUL ****/
__global__ void matMulGridStride(float* A, float* B, float* C, int m, int k, int n) {
    int ROW = blockIdx.x*blockDim.x+threadIdx.x;
    int COL = blockIdx.y*blockDim.y+threadIdx.y;

    int Rstride = blockDim.x*gridDim.x;
    int Cstride = blockDim.y*gridDim.y;    

    for (int r=ROW; r<m; r+=Rstride) {
        for (int c=COL; c<n; c+=Cstride) {        
            float tmpSum = 0.0f;
            for (int i = 0; i <k; ++i) {
                tmpSum += A[(r*k)+i] * B[(i*n)+c];
            }
            C[(r*n)+c] = tmpSum;
        }           
    }   
    return ;
}

/**** GRID-STRIDE SQUARE MATMUL ****/
__global__ void squareMatMulGridStrideKer(float* A, float* B, float* C, int N) {

    int ROW = blockIdx.x*blockDim.x+threadIdx.x;
    int COL = blockIdx.y*blockDim.y+threadIdx.y;

    int Rstride = blockDim.x*gridDim.x;
    int Cstride = blockDim.y*gridDim.y;    

    float tmpSum = 0.0f;
    for (int k=ROW; k<N; k+=Rstride) {
        for (int j=COL; j<N; j+=Cstride) {        
            tmpSum=0;
            for (int i = 0; i < N; i++) {
                tmpSum += A[(k*N)+i] * B[(i*N)+j];
            }
            C[(k*N)+j] = tmpSum;
        }               
    }  
    return ;
}

/**** GRID-STRIDE BLUR BOX ****/
__global__ void blurBoxGridStride(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x;
    int fsize = 5; // Filter size

    for(int i=offset; i<width*height; i+=stride)
    {    
        int x = offset % width;
        int y = (offset-x)/width;

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = ( i +ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[i *3] = output_red/hits;
        output_image[i *3+1] = output_green/hits;
        output_image[i *3+2] = output_blue/hits;
    }
    return ;
}


/*******************
**KERNEL LAUNCHERS**
********************/
#ifdef MATMUL
/**** SQUARE MATMUL ****/
void streamSquareMatMul(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, 
            int n, hipStream_t strm, bool shared)
{
    unsigned int bytesMat = n*n*sizeof(float);
    // H2D memCopy
    gpuErrchk( hipMemcpyAsync(Ad, A, bytesMat, hipMemcpyHostToDevice, strm) );    
    gpuErrchk( hipMemcpyAsync(Bd, B, bytesMat, hipMemcpyHostToDevice, strm) );   
    // Grid and Block setting
    dim3 dimBlock( BLOCK,BLOCK,1 );
    dim3 dimGrid( GRIDx,GRIDx,1 ); 
    // Kernel launch
    if(shared)
        sharedMatMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    else
        squareMatMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, n);
    // D2H memCopy
    gpuErrchk( hipMemcpyAsync( C, Cd, bytesMat, hipMemcpyDeviceToHost, strm) );

    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
    #endif 
}

void squareMatMul(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, int n, bool shared)
{
    unsigned int bytesMat = n*n*sizeof(float);
    // H2D memCopy
    gpuErrchk( hipMemcpy(Ad, A, bytesMat, hipMemcpyHostToDevice) );    
    gpuErrchk( hipMemcpy(Bd, B, bytesMat, hipMemcpyHostToDevice) );   
    // Grid and Block setting 
    dim3 dimBlock( BLOCK,BLOCK,1 );
    dim3 dimGrid( GRIDx,GRIDx,1 ); 
    // Kernel launch
    if(shared)
        sharedMatMulKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, n);
    else
        squareMatMulKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, n);
    // D2H memCopy
    gpuErrchk( hipMemcpy( C, Cd, bytesMat, hipMemcpyDeviceToHost) );

    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
    #endif 
}

/**** NON SQUARE MATMUL ****/
void streamMatMul(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, 
        int m, int k, int n, hipStream_t strm)
{
    unsigned int bytesA = m*k*sizeof(float);
    unsigned int bytesB = k*n*sizeof(float);
    unsigned int bytesC = m*n*sizeof(float);
    // H2D memCopy
    hipMemcpyAsync(Ad, A, bytesA, hipMemcpyHostToDevice, strm);    
    hipMemcpyAsync(Bd, B, bytesB, hipMemcpyHostToDevice, strm);   
    // Grid and Block setting    
    dim3 dimBlock( BLOCK,BLOCK,1 );
    dim3 dimGrid( GRIDx,GRIDy,1 ); 
    // Kernel launch
    matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);
    // D2H memCopy
    hipMemcpyAsync( C, Cd, bytesC, hipMemcpyDeviceToHost, strm);
}


void matMul(float *A, float *B, float *C, float *Ad, float *Bd, float *Cd, 
        int m, int k, int n)
{
    unsigned int bytesA = m*k*sizeof(float);
    unsigned int bytesB = k*n*sizeof(float);
    unsigned int bytesC = m*n*sizeof(float);
    // H2D memCopy
    hipMemcpy(Ad, A, bytesA, hipMemcpyHostToDevice);    
    hipMemcpy(Bd, B, bytesB, hipMemcpyHostToDevice);   
    // Grid and Block setting
    dim3 dimBlock( BLOCK,BLOCK,1 );
    dim3 dimGrid( GRIDx,GRIDy,1 ); 
    // Kernel launch
    matMulKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, m,  k,  n);
    // D2H memCopy
    hipMemcpy( C, Cd, bytesC, hipMemcpyDeviceToHost);
}
#endif



/**** BLURBOX ****/
#ifdef BLURBOX
void streamBlurBoxFilter (unsigned char *in_h, unsigned char *out_h, unsigned char *in_d, unsigned char *out_d, 
                    int width, int height, hipStream_t strm)
{          
    unsigned int size = width*height*3;
    // H2D memCopy
    gpuErrchk( hipMemcpyAsync(in_d, in_h, size, hipMemcpyHostToDevice, strm) );  
    // Grid and Block setting
    #ifdef LOWPAR   
        GRIDx = 1;     
    #else
        GRIDx = (unsigned int)((size+BLOCK-1)/BLOCK);
    #endif
    // Kernel launch
    #ifdef LOWPAR   
        blurBoxGridStride<<<GRIDx, BLOCK, 0, strm>>>(in_d, out_d, width, height); 
    #else
        blurBoxFilterKer<<<GRIDx, BLOCK, 0, strm>>>(in_d, out_d, width, height); 
    #endif
    // D2H memCopy
    gpuErrchk( hipMemcpyAsync( out_h, out_d, size, hipMemcpyDeviceToHost, strm) );
   
    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
    #endif 
}

void blurBoxFilter (unsigned char *in_h, unsigned char *out_h, unsigned char *in_d, unsigned char *out_d, 
                    int width, int height)
{     
    unsigned int size = width*height*3;    
    // H2D memCopy
    gpuErrchk( hipMemcpy(in_d,in_h, size, hipMemcpyHostToDevice) );   
    // Grid and Block setting         
    #ifdef LOWPAR   
        GRIDx = 1;           
    #else
        GRIDx = (unsigned int)((size+BLOCK-1)/BLOCK);
    #endif
    dim3 blockDims( BLOCK,1,1 );   
    dim3 gridDims( GRIDx,1,1 ); 
    // Kernel launch
    #ifdef LOWPAR   
        blurBoxGridStride<<<gridDims, blockDims>>>(in_d, out_d, width, height); 
    #else
        blurBoxFilterKer<<<gridDims, blockDims>>>(in_d, out_d, width, height); 
    #endif
    // D2H memCopy
    gpuErrchk( hipMemcpy( out_h, out_d, size, hipMemcpyDeviceToHost) );
}
#endif