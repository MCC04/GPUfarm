#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <assert.h> 
#include <cstdlib>
#include <algorithm>
#include <ctime>
#include <vector>
#include <future>
#include <iterator>
#include <numeric>

#define HIGH 500.0f
#define LOW -500.0f

#define GRID 1
#define BLOCK 32

//M = iterations; N = size
__global__ void cosKernel(int M, int N, float *x_d, int offset, int *myclocks){    
    int idx = offset+blockIdx.x*blockDim.x + threadIdx.x; 
   
    clock_t start =clock();

    
    for(int i=0;i<N;i+=1){
        for(int j=0;j<M;j+=1)
        {
            x_d[idx+i]=cosf(x_d[idx+i]);  
        }    //x_d[idx]=cosf(x_d[idx]);    
        //

    }
    
    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x]=(int)(end-start);
    return ;
}

// Function to check any CUDA runtime API results
inline hipError_t checkCuda(hipError_t result)
{
    #if defined(DEBUG) || defined(_DEBUG)
        if (result != cudaSuccess) {
        std::cout <<  "CUDA Runtime Error: " << cudaGetErrorString(result)<< std::endl;
        assert(result == cudaSuccess);
        }
    #endif
        return result;
}

int main(int argc, char **argv){
    std::srand(static_cast <unsigned> (time(NULL)));

    
    int gpu_clk=1;
    float msSum=0.0; // elapsed time in milliseconds

    int devId = atoi(argv[1]);
    int K_exec = atoi(argv[2]);
    int M_iter = atoi(argv[3]);
    int N_size = atoi(argv[4]);

    //int BLOCK=N_size;

    float *x=new float[N_size];      
     
    checkCuda(hipDeviceGetAttribute(&gpu_clk, hipDeviceAttributeClockRate, devId));
    hipDeviceProp_t prop;
    checkCuda( hipSetDevice(devId) );
    checkCuda( hipGetDeviceProperties(&prop, devId));
    std::cout<<"Device : "<< prop.name <<std::endl;
    std::cout<<"multiproc num : "<< prop.multiProcessorCount <<std::endl;
    std::cout<<"warp size : "<< prop.warpSize <<std::endl;
    std::cout<<"GPU freq (kHz) : "<< gpu_clk <<std::endl<<std::endl;
    
    std::cout << "Items number \t Host iterations \t Kernel iterations " << std::endl;
    std::cout << N_size<<" \t \t \t " << K_exec<< " \t \t \t " << M_iter << std::endl;


#ifdef CPU
    std::cout<<std::endl<<"##########################" <<std::endl;
    std::cout<<"########## CPU ##########" <<std::endl;
    std::cout<<"##########################" <<std::endl;
    
    float *cosx=new float[N_size];
    
    for(int i=0; i<N_size;i+=1)
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
    
    std::chrono::system_clock::time_point start,end;
    std::vector<double> timeVect;
    for (int r = 0; r < K_exec; ++r) { 

        start=std::chrono::system_clock::now();
        for(int i=0;i<N_size;i+=1)    
            for(int j=0;j<M_iter;j+=1)
                cosx[i]=cos(x[i]);  
                
        end=std::chrono::system_clock::now();

        msSum+=(end-start).count()*1000;
        timeVect.push_back((end - start).count()*1000);
        #if !defined(MEASURES)
            std::cout << std::endl;
            std::cout<< std::endl << "********** ITERATION "<<r<<" **********"<< std::endl;  
            std::cout<<"COSX array : " <<std::endl;  
            for(int j=0; j<N_size;j+=1) 
                std::cout << cosx[j] << ", ";    
            
            std::cout<<std::endl<< "Elapsed time: "<<timeVect[r]<<"ms"<< std::endl;
        #else
            std::cout <<"*"<<r<<"  "<<timeVect[r]; 
        #endif
            
    }
    auto m = std::minmax_element(timeVect.begin(), timeVect.end());
    auto min = m.first;
    auto max = m.second;
    float avg=std::accumulate( timeVect.begin(), timeVect.end(), 0.0)/timeVect.size(); 

    std::cout<<std::endl<<"----"<< *min<<","<<*max<<","<<avg<<std::endl; 

#elif ONE_SM
    std::cout<<std::endl<<"##########################" <<std::endl;
    std::cout<<"########## ONE SM ##########" <<std::endl;
    std::cout<<"##########################" <<std::endl;

    const int bytesSize = N_size*sizeof(float);   
    int* myClock, *myClock_d;     
    float *x_d,*cosx,ms=0.0;
    cosx=new float[N_size];
    myClock=new int[GRID];

    float  rb_wb=0.0;
    //GRID=1;    

    cudaEvent_t startEvent, stopEvent;
    checkCuda( cudaEventCreate(&startEvent) );
    checkCuda( cudaEventCreate(&stopEvent) );

    cudaMalloc(&x_d, bytesSize);
    cudaMalloc(&myClock_d, GRID*sizeof(int));

    memset(x, 0, bytesSize);
    memset(cosx, 0, bytesSize);
    //memset(myClock_d, 0, GRID*sizeof(int));
      
    for(int i=0; i<N_size;i+=1)
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
 
       // checkCuda( cudaEventCreate(&startEvent) );
    //checkCuda( cudaEventCreate(&stopEvent) );
    //cudaEventCreate(&startEvent);
    //cudaEventCreate(&stopEvent);
    for (int r = 0; r < K_exec; ++r) {       
        float tmp=0.0;
        checkCuda( cudaEventRecord(startEvent,0) );

        checkCuda(cudaMemcpy(x_d, x, bytesSize, cudaMemcpyHostToDevice));    

        cosKernel<<<GRID, BLOCK>>>(M_iter, N_size, x_d, 0, myClock_d);

        checkCuda(cudaMemcpy( cosx, x_d, bytesSize, cudaMemcpyDeviceToHost));

        checkCuda(cudaMemcpy( myClock, myClock_d, GRID*sizeof(int), cudaMemcpyDeviceToHost));

        checkCuda( cudaEventRecord(stopEvent, 0) );
        checkCuda( cudaEventSynchronize(stopEvent) );
        checkCuda( cudaEventElapsedTime(&tmp, startEvent, stopEvent) );
        //ms+=tmp;      
        
        #if !defined(MEASURES)
            std::cout<<"COSX array : " <<std::endl;  
            for(int j=0; j<N_size;j+=1) 
                std::cout << cosx[j] << ", ";    
            std::cout << std::endl;
            std::cout<< std::endl << "********** ITERATION "<<r<<" **********"<< std::endl;     
        #endif
        rb_wb=bytesSize*2 + GRID*sizeof(int);        

        #ifdef MEASURES
            std::cout <<"*"<<r<<"," <<myClock[0]/(float)gpu_clk << ","<< myClock[0] << ","<< 
                    tmp<< ","<< (rb_wb/tmp/1e6)<<std::endl; 
        #else
            std::cout<< std::endl <<"-------------------------"<< std::endl; 
            std::cout << "GPU freq (kHz) \t Clk (ms) \t my clk \t event time(ms) "<< std::endl;   
            std::cout << gpu_clk << " \t " << myClock[0]/(float)gpu_clk << " \t "<< myClock[0] <<" \t "<< tmp<<std::endl; 
            std::cout<<std::endl<<"----Effective Bandwidth: "<< (rb_wb/tmp/1e6)<<"GB/s"<<std::endl;        
        #endif
        msSum+=tmp;
    }
    cudaFree(x_d);
#endif

    std::cout<<std::endl<<"----Total Events measures: "<< msSum<<"ms"<<std::endl;


    return 0;
}