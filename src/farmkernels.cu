#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
 
#include <cstdlib>
#include <algorithm>
#include <ctime>
#include <vector>
#include <future>
#include <iterator>
#include <cosFutStr.h>

#define HIGH 500.0f
#define LOW -500.0f


void randomArray(float *x, int n){
    #ifndef MEASURES
            std::cout<<std::endl<< "X ARRAY: "<<std::endl;  
    #endif
    for(int i=0; i<n;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
        #ifndef MEASURES
            std::cout<< x[i] << ", ";  
        #endif
    }
}

/*********
**KERNELS*
**********/
__global__ void emptyKernel(){ return; }

__global__ void cosKernel(int M, int N, float *x_d, int *myclocks, int offset){    
    int idx = offset+blockIdx.x*blockDim.x + threadIdx.x; 
   
    clock_t start =clock();

    if(idx<N){
        for(int j=0; j<M; ++j)
        x_d[idx]=cosf(x_d[idx]);  
    }
    

    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);
    return ;
}

__global__ void cosGridStride(int M, int N, float *x_d, int *myclocks, int offset){    
    int index = offset+blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    clock_t start =clock();
    for (int i = index; i < N; i += stride)
    {
        for(int j=0;j<M;j+=1)
            x_d[i]=cosf(x_d[i]);  
    }
    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);

    return ;
}


/******************
* KERNEL LAUNCERS *
*******************/
float emptyKer(){
    float ms=0;
    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );   

    checkCuda( hipEventRecord(startEvent,0) );
    
    emptyKernel<<<GRID, BLOCK>>>();

    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );

    return ms;    
}



/*void cosKer(my_struct *_xs, float *x_d, int *clocks_d, int chunkBytes,
            hipEvent_t start, hipEvent_t stop, hipStream_t strm)
{    
    checkCuda( hipEventRecord(start,0) );

    checkCuda(hipMemcpyAsync(x_d, _xs->x_vect, chunkBytes, hipMemcpyHostToDevice, strm));
    #ifdef LOWPAR
        //cosGridStride<<<GRID, BLOCK, 0, strm>>>(M_iter, N_size, x_d, clocks_d, 0);
        cosGridStride<<<GRID, BLOCK, 0, strm>>>(M_iter, chunk, x_d, clocks_d, 0);
    #else
        //cosKernel<<<GRID, BLOCK, 0, strm>>>(M_iter, N_size, x_d, clocks_d, 0);
        cosKernel<<<GRID, BLOCK, 0, strm>>>(M_iter, chunk, x_d, clocks_d, 0);
    #endif
    checkCuda(hipMemcpyAsync( _xs->x_vect, x_d, chunkBytes, hipMemcpyDeviceToHost, strm));
    checkCuda(hipMemcpyAsync( _xs->clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost, strm));

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&_xs->eventTime, start, stop) );            
}*/


//FUTURE VECCHIO
/*void cosKer(std::vector<my_struct> &getDatas, int chunk, int bytesSize )
{
    std::vector<std::future<my_struct>> futures;
    int *clocks_d;
    float *x_d;    
    //float *x = new float[N_size];

    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );    

    checkCuda(hipMalloc((void **)&x_d, bytesSize)); 
    checkCuda(hipMalloc((void **)&clocks_d, GRID*sizeof(int)));


    checkCuda( hipEventRecord(startEvent,0) );
    for(int i = 0; i < K_exec; ++i) {
        futures.push_back (std::async(std::launch::async,//std::launch::deferred,
            [&]() { 
                my_struct _xs;
                _xs.clocks=new int[GRID];
                _xs.x_vect=new float[chunk];
                randomArray(_xs.x_vect, chunk);
                //randomArray(&x[i*chunk], chunk);

                //checkCuda( hipEventRecord(startEvent,0) );

                checkCuda(hipMemcpy(x_d, _xs.x_vect, bytesSize, hipMemcpyHostToDevice)); 
                //checkCuda(hipMemcpy(x_d, &x[i*chunk], bytesSize, hipMemcpyHostToDevice)); 

                #ifdef LOWPAR
                    cosGridStride<<<GRID, BLOCK>>>(M_iter, chunk, x_d, clocks_d, 0);
                #else
                    cosKernel<<<GRID, BLOCK>>>(M_iter, chunk, x_d,clocks_d, 0);
                #endif
                              
                checkCuda(hipMemcpy( _xs.x_vect, x_d, bytesSize, hipMemcpyDeviceToHost));
                checkCuda(hipMemcpy(_xs.clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost));

                /*checkCuda( hipEventRecord(stopEvent, 0) );
                checkCuda( hipEventSynchronize(stopEvent) );
                checkCuda( hipEventElapsedTime(&_xs.eventTime, startEvent, stopEvent) );
                _xs.eventTime=0;

                return _xs;
            }));          
    }
    float ms=0.0f;
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    
    std::cout << "EVENT TIME FUTURE: "<< ms<<std::endl;
    
    for(auto &e : futures) 
        getDatas.push_back(e.get());
}*/




void printCos(float *cosx){    
    std::cout<<std::endl<<"COSX array : " <<std::endl;  
    //int chunk = N_size/K_exec;
    for(int j=0; j<N_size;j+=1) 
        std::cout << cosx[j] << ", ";    
    std::cout << std::endl;

      
}




//FUTURE CON STREAM
/*void cosKer(std::vector<my_struct> &getDatas, int chunk, int bytesSize )
{
    std::vector<std::future<my_struct>> futures;
    int *clocks_d;
    float *x_d;    
    float *x = new float[N_size];
    float *cosx = new float[N_size];
    float *clockss = new float[K_exec*GRID];

    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );    

    checkCuda( hipMalloc((void **)&x_d, N_size*sizeof(float)) ); 
    checkCuda( hipMalloc((void **)&clocks_d, GRID*K_exec*sizeof(int)) );


    checkCuda( hipEventRecord(startEvent,0) );

    randomArray(x, N_size);
    //hipStream_t str1;
    //checkCuda( hipStreamCreate(&str1) );
    hipStream_t *stream=streamCreate(3);
    for(int i = 0; i < K_exec; ++i) {
        
     
        futures.push_back (
         //auto myFut=   
         std::async(std::launch::async,//std::launch::deferred,//
            [x,x_d,clocks_d,chunk,stream](int i ) { 
                my_struct _xs;
                _xs.clocks=new int[GRID];
                _xs.x_vect=new float[chunk];
                int k = i%3;
                //randomArray(_xs.x_vect, chunk);
                //randomArray(&x[i*chunk], chunk);

                //checkCuda( hipEventRecord(startEvent,0) );

                //checkCuda( hipMemcpy(&x_d[i*chunk], &x[i*chunk], bytesSize, hipMemcpyHostToDevice) ); 
                std::cout <<i<<" - going to memcpy x in H2D..."<<std::endl;
                checkCuda( hipMemcpyAsync(&x_d[i*chunk], &x[i*chunk], chunk*sizeof(float), hipMemcpyHostToDevice, stream[k]) );          
                std::cout << i<<"- done memcpy x in H2D!"<<std::endl;
                ////checkCuda(hipMemcpy(x_d, &x[i*chunk], bytesSize, hipMemcpyHostToDevice)); 

                #ifdef LOWPAR
                    cosGridStride<<<GRID, BLOCK,0,stream[k]>>>(M_iter, chunk, &x_d[i*chunk], &clocks_d[i*chunk], 0);
                #else
                    std::cout << i<<"- kernel launch..."<<std::endl;

                    cosKernel<<<GRID, BLOCK,0,stream[k]>>>(M_iter, chunk, &x_d[i*chunk],&clocks_d[i*chunk], 0);
                    hipDeviceSynchronize();
                    std::cout << i<<"- kernel end!"<<std::endl;

                #endif
                              
                //checkCuda( hipMemcpy( _xs.x_vect, &x_d[i*chunk], chunk*sizeof(float), hipMemcpyDeviceToHost) );
                //checkCuda( hipMemcpy(_xs.clocks, &clocks_d[i*GRID], GRID*sizeof(int), hipMemcpyDeviceToHost) );
                std::cout << i<<"- going to memcpy in D2H..."<<std::endl;

                checkCuda( hipMemcpyAsync(_xs.x_vect, &x_d[i*chunk], chunk*sizeof(float), hipMemcpyDeviceToHost, stream[k]) );
                checkCuda( hipMemcpyAsync(_xs.clocks, &clocks_d[i*GRID], GRID*sizeof(int), hipMemcpyDeviceToHost, stream[k]) );
                std::cout << i<<"- done memcpy H2D..."<<std::endl;

                //checkCuda( hipEventRecord(stopEvent, 0) );
                //checkCuda( hipEventSynchronize(stopEvent) );
                //checkCuda( hipEventElapsedTime(&_xs.eventTime, startEvent, stopEvent) );
                _xs.eventTime=0;

                return _xs;
            },i));  

            //auto obj=myFut.get(); 

        
    }
    float ms=0.0f;

    
    for(auto &e : futures) 
            getDatas.push_back(e.get()); 
    
       //printCos(cosx);
        
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );

    std::cout << "EVENT TIME FUTURE: "<< ms<<std::endl;
streamDestroy(stream,3);
}*/



//NB: launch deferred funziona sempre (sia memcpy, sia memcpyasync) perché non crea overlapping
//facendo la lambda quando chiamo get, non so esattamente perché, ma
//non ho più qualcosa che somigli a chiamate asincrone, ma diventa tutto
//seriale. Esempio: h2d, ker0, d2h -- h2d, ker1, d2h ....



//Le memcpy danno problemi perché quando uso quelle con async,
//partono tutte le memcpy h2d e dopo tutti i kernel e le d2h.
//questo probabilmente collegato al fatto che sono sincrone e
//quindi nel frattempo che viene aspettata la copia parte la memcpy
//successiva. Però poi quando chiamo i kernel tutti insieme succede un casino
//col mapping della memoria, hp eccezioni di accesso illegale alla memoria.
//Questo probabilmente è dovuto al fatto che es sono state fatte molte 
//memcpy, arriva il kernelO e la sua memoria è stata "manomessa" da 
//altre memcopy ---> però c'è da dire che in teoria userei porzioni diverse
//dell'array x_d allocato in device.
//magari sbaglio qualcosa nelle memcpy?
//fatto sta che non è un'ottimo risultato in quanto a overlapping in ogni caso
//per questo credo sia meglio fare con memcpyasync

void cosKer(std::vector<my_struct> &getDatas, int chunk, int bytesSize )
{
    std::vector<std::future<my_struct>> futures;
    int *clocks_d;
    float *x_d; 

    float *x = new float[N_size];
    //checkCuda( hipHostMalloc((void **)&x, N_size*sizeof(float)) ); //pinned x
    float *cosx = new float[N_size];
    //checkCuda( hipHostMalloc((void **)&cosx, N_size*sizeof(float)) ); //pinned x
    float *clockss = new float[K_exec*GRID];
    //checkCuda( hipHostMalloc((void **)&clockss, K_exec*GRID*sizeof(float)) ); //pinned x


    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );    

    checkCuda( hipMalloc((void **)&x_d, N_size*sizeof(float)) ); 
    checkCuda( hipMalloc((void **)&clocks_d, GRID*K_exec*sizeof(int)) );


    checkCuda( hipEventRecord(startEvent,0) );

    randomArray(x, N_size);
    //hipStream_t str1;
    //checkCuda( hipStreamCreate(&str1) );
    hipStream_t *stream=streamCreate(3);
    for(int i = 0; i < K_exec; ++i) {
        
     
        futures.push_back (
         //auto myFut=   
         std::async(std::launch::deferred,//std::launch::async,//
            //[x,x_d,clocks_d,chunk](int i ) { 
             [x,x_d,clocks_d,chunk,stream](int i ) { 

                my_struct _xs;
                _xs.clocks=new int[GRID];
                _xs.x_vect=new float[chunk];
                int k = i%3;
                //randomArray(_xs.x_vect, chunk);
                //randomArray(&x[i*chunk], chunk);

                //checkCuda( hipEventRecord(startEvent,0) );
                std::cout <<i<<" - going to memcpy x in H2D..."<<std::endl;
                //checkCuda( hipMemcpy(&x_d[i*chunk], &x[i*chunk], chunk*sizeof(float), hipMemcpyHostToDevice) ); 
                
               checkCuda( hipMemcpyAsync(&x_d[i*chunk], &x[i*chunk], chunk*sizeof(float), hipMemcpyHostToDevice, stream[k]) );          
                std::cout << i<<"- done memcpy x in H2D!"<<std::endl;
                ////checkCuda(hipMemcpy(x_d, &x[i*chunk], bytesSize, hipMemcpyHostToDevice)); 

               // #ifdef LOWPAR
                    
                     std::cout << i<<"- kernel launch..."<<std::endl;
                     cosGridStride<<<GRID, BLOCK,0,stream[k]>>>(M_iter, chunk, &x_d[i*chunk], &clocks_d[i*chunk], 0);
                     //cosGridStride<<<GRID, BLOCK>>>(M_iter, chunk, &x_d[i*chunk], &clocks_d[i*chunk], 0);
               // #else
                   
                 //   cosKernel<<<GRID, BLOCK>>>(M_iter, chunk, &x_d[i*chunk],&clocks_d[i*chunk], 0);
                    //cosKernel<<<GRID, BLOCK,0,stream[k]>>>(M_iter, chunk, &x_d[i*chunk],&clocks_d[i*chunk], 0);
                    //hipDeviceSynchronize();
                    std::cout << i<<"- kernel end!"<<std::endl;

              //  #endif
                              
                
                std::cout << i<<"- going to memcpy in D2H..."<<std::endl;
                //checkCuda( hipMemcpy( _xs.x_vect, &x_d[i*chunk], chunk*sizeof(float), hipMemcpyDeviceToHost) );
                //checkCuda( hipMemcpy(_xs.clocks, &clocks_d[i*GRID], GRID*sizeof(int), hipMemcpyDeviceToHost) );
                checkCuda( hipMemcpyAsync(_xs.x_vect, &x_d[i*chunk], chunk*sizeof(float), hipMemcpyDeviceToHost, stream[k]) );
                checkCuda( hipMemcpyAsync(_xs.clocks, &clocks_d[i*GRID], GRID*sizeof(int), hipMemcpyDeviceToHost, stream[k]) );
                std::cout << i<<"- done memcpy H2D..."<<std::endl;

                /*checkCuda( hipEventRecord(stopEvent, 0) );
                checkCuda( hipEventSynchronize(stopEvent) );
                checkCuda( hipEventElapsedTime(&_xs.eventTime, startEvent, stopEvent) );*/
                _xs.eventTime=0;

                return _xs;
            },i));  

            //auto obj=myFut.get(); 

        
    }
    float ms=0.0f;

    
    for(auto &e : futures) 
            getDatas.push_back(e.get()); 
    
       //printCos(cosx);
        
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );

    std::cout << "EVENT TIME FUTURE: "<< ms<<std::endl;
    streamDestroy(stream,3);

        hipHostFree(x);
        hipHostFree(cosx);
        hipHostFree(clockss);

    hipFree(x_d);
    hipFree(clocks_d);
}




















void cosKerStream(
    int m, int chunk,
    float *x, int *clocks, 
    int offset, hipStream_t strm)
{
       /* #ifdef LOWPAR
            cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, n, x, clocks, offset);
        #else
            cosKernel<<<GRID, BLOCK, offset, strm>>>(m, n, x, clocks, offset);
        #endif*/
       // #ifdef LOWPAR
            cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, x, clocks, offset);
       // #else
         //   cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, x, clocks, offset);
        //#endif
}

float  cosKerStream(
    hipEvent_t start, hipEvent_t stop,
    int m, int chunk,//int n,
    float *x, float *cosx,  int *clocks, 
    int offset, hipStream_t strm)
{
    float ms;  
    //randomArray(x, n);
    //memcpy(cosx,x,N_size);
    randomArray(x,chunk);
    memcpy(cosx,x,chunk);
    
    checkCuda( hipEventRecord(start,0) );

    //#ifdef STRIDE
    #ifdef LOWPAR
        //cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #else
        //cosKernel<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
     
    return ms;
}
