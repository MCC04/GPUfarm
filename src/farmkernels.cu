#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
 
#include <algorithm>
//#include <ctime>
//#include <vector>
//#include <future>
//#include <iterator>
#include <cosFutStr.h>











#define HIGH 500.0f
#define LOW -500.0f


void randomArray(float *x, int n){
    /*#ifndef MEASURES
            std::cout<<std::endl<< "X ARRAY: "<<std::endl;  
    #endif*/
    for(int i=0; i<n;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
       /* #ifndef MEASURES
            std::cout<< x[i] << ", ";  
        #endif*/
    }
}

/*********
**KERNELS*
**********/
#ifdef EMPTY
__global__ void emptyKernel(){ return; }
#endif

/**** GRID-STRIDE COS KERNEL ****/ 
__global__ void cosKernel(int M, int N, float *x_d, int *myclocks, int offset){    
    int idx = offset+blockIdx.x*blockDim.x + threadIdx.x; 
   
    if(idx<N){
        clock_t start =clock();

        for(int j=0; j<M; ++j)
            x_d[idx]=cosf(x_d[idx]);  

        clock_t end=clock();

        if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);
    }
    return ;
}


/**** GRID-STRIDE COS KERNEL ****/ 
__global__ void cosGridStride(int M, int N, float *x_d, int *myclocks, int offset){    
    int index = offset+blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    clock_t start =clock();
    for (int i = index; i < N; i += stride)
    {
        for(int j=0;j<M;j+=1)
            x_d[i]=cosf(x_d[i]);  
    }
    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);

    return ;
}


/******************
* KERNEL LAUNCERS *
*******************/
/*void futCheck(int i, std::string mess){
    #ifndef MEASURES
        std::cout <<i<<mess<<std::endl;
    #endif  
}*/

/**** FUTURE ****/
#ifdef FUTURE
std::vector<std::future<hostData_t>> 
    cosKerFuture(int M, int chunk, hostData_t output, float *x, float *x_d, int *clocks_d, hipStream_t *strm, int nStreams, int offset)
{
    std::vector<std::future<hostData_t>> futures; 
    //int strBytes = ;

    for(int i = 0; i < K_exec; ++i) {
        int k = i%nStreams;
        randomArray(x+i*chunk,chunk);
  
     
        futures.push_back (std::async(std::launch::async,//std::launch::deferred,//       
                [M, chunk, output, x_d, clocks_d, offset,i] (float * x, hipStream_t strm, int strBytes) {
                
                std::cout <<i<<"- going to memcpy x in H2D..."<<std::endl;
              
                gpuErrchk( hipMemcpyAsync(x_d, x, strBytes, hipMemcpyHostToDevice, strm) ); //gpuErrchk( hipMemcpy(&x_d[i*chunk], &x[i*chunk], chunk*sizeof(float), hipMemcpyHostToDevice) );          

                std::cout <<i<<"- done memcpy x in H2D!"<<std::endl;



                #ifdef LOWPAR
                    std::cout <<i<<"- kernel launch..."<<std::endl;

                    cosGridStride<<<GRID, BLOCK,0,strm>>>(M, chunk, x_d, clocks_d, offset); //cosGridStride<<<GRID, BLOCK>>>(M_iter, chunk, &x_d[i*chunk], &clocks_d[i*chunk], 0);

                    std::cout <<i<<"- kernel end!"<<std::endl;
                #else

                    std::cout <<i<<"- kernel launch..."<<std::endl;

                    cosKernel<<<GRID, BLOCK,0,strm>>>(M, chunk, x_d, clocks_d, offset); //cosKernel<<<GRID, BLOCK>>>(M_iter, chunk, &x_d[i*chunk],&clocks_d[i*chunk], 0);

                    std::cout <<i<<"- kernel end!"<<std::endl;

                #endif



                std::cout <<i<<"- going to memcpy x in D2H..."<<std::endl;

                gpuErrchk( hipMemcpyAsync(output.x, x_d, strBytes, hipMemcpyDeviceToHost, strm) ); //gpuErrchk( hipMemcpy( output.x_vect, &x_d[i*chunk], chunk*sizeof(float), hipMemcpyDeviceToHost) );
                gpuErrchk( hipMemcpyAsync(output.clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost, strm) ); //gpuErrchk( hipMemcpy(output.clocks, &clocks_d[i*GRID], GRID*sizeof(int), hipMemcpyDeviceToHost) );


                std::cout <<i<<"- done memcpy x in D2H!"<<std::endl;

                return output;
            }, x+(i*chunk), strm[k], chunk*sizeof(float) ));       
    }
    return futures;
}
#endif


/**** STREAM ****/
#ifdef STREAM

void cosKerStream(int m, int chunk, float *x, float *cosx, float *x_d, int *clocks, int *clocks_d, hipStream_t strm, int strBytes, int offset)
{    
    gpuErrchk( hipMemcpyAsync(x_d, x, strBytes, hipMemcpyHostToDevice, strm) ); 
    #ifdef LOWPAR
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, x_d, clocks_d, offset);
    #else
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, x_d, clocks_d, offset);
    #endif   

    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    #endif   
    gpuErrchk( hipMemcpyAsync( cosx, x_d, strBytes, hipMemcpyDeviceToHost, strm) );
    gpuErrchk( hipMemcpyAsync( clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost, strm) );

    #ifndef MEASURES
        printClocks(clocks,GRID);
    #endif         
}
#endif


/**** STREAM MANAGED ****/
#ifdef MANAGED
void  cosKerStream(
    //hipEvent_t start, hipEvent_t stop,
    int m, int chunk,//int n,
    float *x, float *cosx,  int *clocks, 
    int offset, hipStream_t strm)
{
    float ms;  
    //randomArray(x, n);
    //memcpy(cosx,x,N_size);
    randomArray(x,chunk);
    memcpy(cosx,x,chunk);
    
    //gpuErrchk( hipEventRecord(start,0) );

    //#ifdef STRIDE
    #ifdef LOWPAR
        //cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #else
        //cosKernel<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #endif

    /*gpuErrchk( hipEventRecord(stop, 0) );
    gpuErrchk( hipEventSynchronize(stop) );
    gpuErrchk( hipEventElapsedTime(&ms, start, stop) );*/
     
   // return ms;
}
#endif


/**** EMPTY ****/
#ifdef EMPTY
float emptyKer(){
    float ms=0;
    hipEvent_t startEvent, stopEvent;
    gpuErrchk( hipEventCreate(&startEvent) );
    gpuErrchk( hipEventCreate(&stopEvent) );   

    gpuErrchk( hipEventRecord(startEvent,0) );
    
    emptyKernel<<<GRID, BLOCK>>>();

    gpuErrchk( hipEventRecord(stopEvent, 0) );
    gpuErrchk( hipEventSynchronize(stopEvent) );
    gpuErrchk( hipEventElapsedTime(&ms, startEvent, stopEvent) );

    return ms;    
}
#endif