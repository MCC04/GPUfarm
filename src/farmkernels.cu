#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
 
#include <cstdlib>
#include <algorithm>
#include <ctime>
#include <vector>
#include <future>
#include <iterator>
#include <cosFutStr.h>

#define HIGH 500.0f
#define LOW -500.0f


void randomArray(float *x, int n){
    #ifndef MEASURES
            std::cout<<std::endl<< "X ARRAY: "<<std::endl;  
    #endif
    for(int i=0; i<n;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
        #ifndef MEASURES
            std::cout<< x[i] << ", ";  
        #endif
    }
}

/*********
**KERNELS*
**********/
__global__ void emptyKernel(){ return; }

__global__ void cosKernel(int M, int N, float *x_d, int *myclocks, int offset){    
    int idx = offset+blockIdx.x*blockDim.x + threadIdx.x; 
   
    clock_t start =clock();

    for(int j=0;j<M;j+=1)
        x_d[idx]=cosf(x_d[idx]);  

    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);
    return ;
}

__global__ void cosGridStride(int M, int N, float *x_d, int *myclocks, int offset){    
    int index = offset+blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    clock_t start =clock();
    for (int i = index; i < N; i += stride)
    {
        for(int j=0;j<M;j+=1)
            x_d[i]=cosf(x_d[i]);  
    }
    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);

    return ;
}


/******************
* KERNEL LAUNCERS *
*******************/
float emptyKer(){
    float ms=0;
    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );   

    checkCuda( hipEventRecord(startEvent,0) );
    
    emptyKernel<<<GRID, BLOCK>>>();

    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );

    return ms;    
}



/*void cosKer(my_struct *_xs, float *x_d, int *clocks_d, int chunkBytes,
            hipEvent_t start, hipEvent_t stop, hipStream_t strm)
{    
    checkCuda( hipEventRecord(start,0) );

    checkCuda(hipMemcpyAsync(x_d, _xs->x_vect, chunkBytes, hipMemcpyHostToDevice, strm));
    #ifdef LOWPAR
        //cosGridStride<<<GRID, BLOCK, 0, strm>>>(M_iter, N_size, x_d, clocks_d, 0);
        cosGridStride<<<GRID, BLOCK, 0, strm>>>(M_iter, chunk, x_d, clocks_d, 0);
    #else
        //cosKernel<<<GRID, BLOCK, 0, strm>>>(M_iter, N_size, x_d, clocks_d, 0);
        cosKernel<<<GRID, BLOCK, 0, strm>>>(M_iter, chunk, x_d, clocks_d, 0);
    #endif
    checkCuda(hipMemcpyAsync( _xs->x_vect, x_d, chunkBytes, hipMemcpyDeviceToHost, strm));
    checkCuda(hipMemcpyAsync( _xs->clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost, strm));

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&_xs->eventTime, start, stop) );            
}*/



void cosKer(std::vector<my_struct> &getDatas, int chunk, int bytesSize )
{
    std::vector<std::future<my_struct>> futures;
    int *clocks_d;
    float *x_d;    
    //float *x = new float[N_size];

    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );    

    checkCuda(hipMalloc((void **)&x_d, bytesSize)); 
    checkCuda(hipMalloc((void **)&clocks_d, GRID*sizeof(int)));

    for(int i = 0; i < K_exec; ++i) {
        futures.push_back (std::async(std::launch::deferred,
            [&]() { 
                my_struct _xs;
                _xs.clocks=new int[GRID];
                _xs.x_vect=new float[chunk];
                //randomArray(_xs.x_vect, chunk);
                randomArray(&x[i*chunk], chunk);

                checkCuda( hipEventRecord(startEvent,0) );

                //checkCuda(hipMemcpy(x_d, _xs.x_vect, bytesSize, hipMemcpyHostToDevice)); 
                checkCuda(hipMemcpy(x_d, &x[i*chunk], bytesSize, hipMemcpyHostToDevice)); 

                #ifdef LOWPAR
                    cosGridStride<<<GRID, BLOCK>>>(M_iter, chunk, x_d, clocks_d, 0);
                #else
                    cosKernel<<<GRID, BLOCK>>>(M_iter, chunk, x_d,clocks_d, 0);
                #endif
                              
                checkCuda(hipMemcpy( _xs.x_vect, x_d, bytesSize, hipMemcpyDeviceToHost));
                checkCuda(hipMemcpy(_xs.clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost));

                checkCuda( hipEventRecord(stopEvent, 0) );
                checkCuda( hipEventSynchronize(stopEvent) );
                checkCuda( hipEventElapsedTime(&_xs.eventTime, startEvent, stopEvent) );

                return _xs;
            }));          
    }
    for(auto &e : futures) 
        getDatas.push_back(e.get());
}

void cosKerStream(
    int m, int chunk,
    float *x, int *clocks, 
    int offset, hipStream_t strm)
{
       /* #ifdef LOWPAR
            cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, n, x, clocks, offset);
        #else
            cosKernel<<<GRID, BLOCK, offset, strm>>>(m, n, x, clocks, offset);
        #endif*/
        #ifdef LOWPAR
            cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, x, clocks, offset);
        #else
            cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, x, clocks, offset);
        #endif
}

float  cosKerStream(
    hipEvent_t start, hipEvent_t stop,
    int m, int chunk,//int n,
    float *x, float *cosx,  int *clocks, 
    int offset, hipStream_t strm)
{
    float ms;  
    //randomArray(x, n);
    //memcpy(cosx,x,N_size);
    randomArray(x,chunk);
    memcpy(cosx,x,chunk);
    
    checkCuda( hipEventRecord(start,0) );

    //#ifdef STRIDE
    #ifdef LOWPAR
        //cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #else
        //cosKernel<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
     
    return ms;
}
