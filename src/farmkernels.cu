#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h> 
#include <algorithm>

#include <cosFutStr.h>

#define HIGH 500.0f
#define LOW -500.0f

void randomArray(float *x, int n){
    for(int i=0; i<n;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
    }
}

/*********
**KERNELS*
**********/
#ifdef EMPTY
__global__ void emptyKernel(){ return; }
#endif

/****  COS KERNEL ****/ 
__global__ void cosKernel(int M, int N, float *x_d, int *myclocks, int offset){    
    int idx = offset+blockIdx.x*blockDim.x + threadIdx.x; 
   
    if(idx<N){
        clock_t start =clock();

        for(int j=0; j<M; ++j)
            x_d[idx]=cosf(x_d[idx]);  

        clock_t end=clock();

      //  if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);
    }
    return ;
}


/**** GRID-STRIDE COS KERNEL ****/ 
__global__ void cosGridStride(int M, int N, float *x_d, int *myclocks, int offset){    
    int index = offset+blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    clock_t start =clock();
    for (int i = index; i < N; i += stride)
    {
        for(int j=0;j<M;j+=1)
            x_d[i]=cosf(x_d[i]);  
    }
    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);

    return ;
}



/******************
* KERNEL LAUNCERS *
*******************/

/**** FUTURE ****/
#ifdef FUTURE
std::vector<std::future<hostData_t>> 
    cosKerFuture(int M, int chunk, hostData_t output, float *x, float *x_d, int *clocks_d, hipStream_t *strm, int nStreams, int offset)
{
    std::vector<std::future<hostData_t>> futures; 

    for(int i = 0; i < K_exec; ++i) {
        const int k = i%nStreams;
        randomArray(x+i*chunk,chunk);

        const int strOffs = k*chunk;
        float *p_xd = x_d+strOffs;
        float *p_x = x+(i*chunk);
        int *p_clocks= clocks_d+(k*GRID);

        futures.push_back (std::async(std::launch::async,     
            [=] (hipStream_t strm, int strBytes) {
            //[M, chunk, p_xd, p_clocks, output, offset,i]

            //std::cout <<i<<"- going to memcpy x in H2D..."<<std::endl;        
            hipMemcpy(p_xd, p_x, strBytes, hipMemcpyHostToDevice);        
            //std::cout <<i<<"- done memcpy x in H2D!"<<std::endl;

            #ifdef LOWPAR
                //std::cout <<i<<"- kernel launch..."<<std::endl;
                cosGridStride<<<GRID, BLOCK,0,strm>>>(M, chunk, p_xd, p_clocks, offset); 
                //std::cout <<i<<"- kernel end!"<<std::endl;
            #else
                //  std::cout <<i<<"- kernel launch..."<<std::endl;
                cosKernel<<<GRID, BLOCK,0,strm>>>(M, chunk, p_xd, p_clocks, offset); 
                //std::cout <<i<<"- kernel end!"<<std::endl;
            #endif

            //std::cout <<i<<"- going to memcpy x in D2H..."<<std::endl;
            hipMemcpy(output.x, p_xd, strBytes, hipMemcpyDeviceToHost) ; 
            hipMemcpy(output.clocks, p_clocks, GRID*sizeof(int), hipMemcpyDeviceToHost) ; 
            //std::cout <<i<<"- done memcpy x in D2H!"<<std::endl;

            return output;
        }, strm[k], chunk*sizeof(float) ));       


        /*futures.push_back (std::async(std::launch::async,//std::launch::deferred,//       
            [M, chunk, x_d, clocks_d, output, offset,i] (float * x, hipStream_t strm, int strBytes) {
            
            //std::cout <<i<<"- going to memcpy x in H2D..."<<std::endl;
            
            
            gpuErrchk( hipMemcpyAsync(x_d, x, strBytes, hipMemcpyHostToDevice, strm) ); //gpuErrchk( hipMemcpy(&x_d[i*chunk], &x[i*chunk], chunk*sizeof(float), hipMemcpyHostToDevice) );          

            //std::cout <<i<<"- done memcpy x in H2D!"<<std::endl;



            #ifdef LOWPAR
                //std::cout <<i<<"- kernel launch..."<<std::endl;

                cosGridStride<<<GRID, BLOCK,0,strm>>>(M, chunk, x_d, clocks_d, offset); //cosGridStride<<<GRID, BLOCK>>>(M_iter, chunk, &x_d[i*chunk], &clocks_d[i*chunk], 0);

                //std::cout <<i<<"- kernel end!"<<std::endl;
            #else

                //  std::cout <<i<<"- kernel launch..."<<std::endl;

                cosKernel<<<GRID, BLOCK,0,strm>>>(M, chunk, x_d, clocks_d, offset); //cosKernel<<<GRID, BLOCK>>>(M_iter, chunk, &x_d[i*chunk],&clocks_d[i*chunk], 0);

                //std::cout <<i<<"- kernel end!"<<std::endl;

            #endif

            //std::cout <<i<<"- going to memcpy x in D2H..."<<std::endl;

            gpuErrchk( hipMemcpyAsync(output.x, x_d, strBytes, hipMemcpyDeviceToHost, strm) ); //gpuErrchk( hipMemcpy( output.x_vect, &x_d[i*chunk], chunk*sizeof(float), hipMemcpyDeviceToHost) );
            gpuErrchk( hipMemcpyAsync(output.clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost, strm) ); //gpuErrchk( hipMemcpy(output.clocks, &clocks_d[i*GRID], GRID*sizeof(int), hipMemcpyDeviceToHost) );

            hipStreamSynchronize(strm);
            //std::cout <<i<<"- done memcpy x in D2H!"<<std::endl;

            return output;
        }, x+(i*chunk), strm[k], chunk*sizeof(float) ));*/  
    }
    return futures;
}
#endif





void optimalCosKer( int m, int chunk, float *x, float *cosx, float *x_d, int *clocks, int *clocks_d, hipStream_t strm, int strBytes, int offset){
    int blockSize;   // The launch configurator returned block size 
    int minGridSize; // The minimum grid size needed to achieve the 
                    // maximum occupancy for a full device launch 
    int gridSize;    // The actual grid size needed, based on input size 

    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, cosKernel, 0, 0); 
    // Round up according to array size 
    gridSize = (chunk + blockSize - 1) / blockSize; 

    gpuErrchk( hipMemcpyAsync(x_d, x, strBytes, hipMemcpyHostToDevice, strm) ); 

    cosKernel<<< gridSize, blockSize,offset,strm >>>(m, chunk, x_d, clocks_d, offset);

    gpuErrchk( hipMemcpyAsync( cosx, x_d, strBytes, hipMemcpyDeviceToHost, strm) );
    gpuErrchk( hipMemcpyAsync( clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost, strm) );

    //hipDeviceSynchronize(); 

    // calculate theoretical occupancy
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, cosKernel, blockSize, 0);

    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);
#ifndef MEASURES
    float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / 
                    (float)(props.maxThreadsPerMultiProcessor / 
                            props.warpSize);

    std::cout << "Launched blockSize: " << blockSize<< std::endl
              << "Min Grid Size: " << minGridSize << std::endl
              << "Launched Grid Size: " << gridSize << std::endl
              << "Max active blocks: " << maxActiveBlocks<< std::endl
              << "Theoretical occupancy:" << occupancy << std::endl;
              
  #endif
}


//DATAPAR
float optimalCosKer( int m, int n, float *x, float *cosx, float *x_d, int *clocks, int *clocks_d){
    int blockSize;   // The launch configurator returned block size 
    int minGridSize; // The minimum grid size needed to achieve the 
                    // maximum occupancy for a full device launch 
    int gridSize;    // The actual grid size needed, based on input size 
    hipEvent_t startEvent, stopEvent;

    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, cosKernel, 0, 0); 
    // Round up according to array size 
    gridSize = (n + blockSize - 1) / blockSize; 

    BLOCK=blockSize;
    GRID=gridSize;

    clocks = (int *) malloc(GRID*sizeof(int));

    gpuErrchk( hipMalloc((void**)&clocks_d, GRID*sizeof(int)) );  

    createAndStartEvent(&startEvent, &stopEvent);   
    gpuErrchk( hipMemcpy(x_d, x, n*sizeof(float), hipMemcpyHostToDevice) ); 

    cosKernel<<<gridSize, blockSize>>>(m, n, x_d, clocks_d, 0);

    gpuErrchk( hipMemcpy( cosx, x_d, n*sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy( clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost) );

    gpuErrchk( hipPeekAtLastError() );

    hipDeviceSynchronize();

    float ms = endEvent(&startEvent, &stopEvent);
    // calculate theoretical occupancy
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, cosKernel, blockSize, 0);

    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);
    #ifndef MEASURES
        float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / 
                        (float)(props.maxThreadsPerMultiProcessor / 
                                props.warpSize);

        std::cout << "Launched blockSize: " << blockSize<< std::endl
                << "Min Grid Size: " << minGridSize << std::endl
                << "Launched Grid Size: " << gridSize << std::endl
                << "Max active blocks: " << maxActiveBlocks<< std::endl
                << "Theoretical occupancy:" << occupancy << std::endl;
   #endif           
  
return ms;


}

/**** STREAM ****/
#ifdef STREAM
void cosKerStream(int m, int chunk, float *x, float *cosx, float *x_d, int *clocks, int *clocks_d, hipStream_t strm, int strBytes, int offset)
{     
    gpuErrchk( hipMemcpyAsync(x_d, x, strBytes, hipMemcpyHostToDevice, strm) ); 

    cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, x_d, clocks_d, offset);
 
    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    #endif   
    gpuErrchk( hipMemcpyAsync( cosx, x_d, strBytes, hipMemcpyDeviceToHost, strm) );
    gpuErrchk( hipMemcpyAsync( clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost, strm) );   
}


void cosKer(int m, int chunk, float *x, float *cosx, float *x_d, int *clocks, int *clocks_d)
{   
    int xBytes = chunk*sizeof(float);
    int clockBytes = GRID*sizeof(int);
    
    gpuErrchk( hipMemcpy(x_d, x, xBytes, hipMemcpyHostToDevice) ); 
    #ifdef LOWPAR
        cosGridStride<<<GRID, BLOCK>>>(m, chunk, x_d, clocks_d, 0);
    #else
        cosKernel<<<GRID, BLOCK>>>(m, chunk, x_d, clocks_d, 0);
    #endif   

    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    #endif   
    gpuErrchk( hipMemcpy( cosx, x_d, xBytes, hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy( clocks, clocks_d, clockBytes, hipMemcpyDeviceToHost) ); 
}

#endif




/**** STREAM MANAGED ****/
#ifdef MANAGED
void  cosKerStream(int m, int chunk,//int n,
    float *x, float *cosx,  int *clocks, int offset, hipStream_t strm)
{
    randomArray(x,chunk);
    memcpy(cosx,x,chunk);

    #ifdef LOWPAR
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #else
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #endif

    hipStreamSynchronize(strm);
}
#endif


/**** EMPTY ****/
#ifdef EMPTY
float emptyKer(){
    float ms=0;
    hipEvent_t startEvent, stopEvent;
    gpuErrchk( hipEventCreate(&startEvent) );
    gpuErrchk( hipEventCreate(&stopEvent) );   

    gpuErrchk( hipEventRecord(startEvent,0) );
    
    emptyKernel<<<GRID, BLOCK>>>();

    gpuErrchk( hipEventRecord(stopEvent, 0) );
    gpuErrchk( hipEventSynchronize(stopEvent) );
    gpuErrchk( hipEventElapsedTime(&ms, startEvent, stopEvent) );

    return ms;    
}
#endif