#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h> 
#include <algorithm>

#include <cosFutStr.h>

#define HIGH 500.0f
#define LOW -500.0f

void randomArray(float *x, int n){
    for(int i=0; i<n;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
    }
}

/*********
**KERNELS*
**********/
/****  COS KERNEL ****/ 
__global__ void cosKernel(int M, int N, float *x_d){    
    int idx = blockIdx.x*blockDim.x + threadIdx.x; 
   
    if(idx<N){
        float x = x_d[idx];
        for(int j=0; j<M; ++j)
            x=cosf(x);  

        x_d[idx] = x;
    }
    return ;
}

/**** GRID-STRIDE COS KERNEL ****/ 
__global__ void cosGridStride(int M, int N, float *x_d){    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float x;
    for (int i = index; i < N; i += stride)
    {
        x = x_d[i];
        for(int j=0;j<M;j+=1)
            x=cosf(x);  

        x_d[i] = x;
    }
    return ;
}


/******************
* KERNEL LAUNCERS *
*******************/

/**** DATAPAR ****/

#ifdef DATAPAR
float optimalCosKer( int m, int n, float *x, float *cosx, float *x_d){
    int blockSize;   // The launch configurator returned block size 
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    hipEvent_t startEvent, stopEvent;

    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, cosKernel, 0, 0); 
    
    GRID = (n + blockSize - 1) / blockSize; // Round up according to array size 
    BLOCK=blockSize;
    // Events creation and start
    createAndStartEvent(&startEvent, &stopEvent);  
    // H2D mem copy 
    gpuErrchk( hipMemcpy(x_d, x, n*sizeof(float), hipMemcpyHostToDevice) ); 
    // Kernel call
    cosKernel<<<GRID, BLOCK>>>(m, n, x_d);
    // D2H mem copy 
    gpuErrchk( hipMemcpy( cosx, x_d, n*sizeof(float), hipMemcpyDeviceToHost) );
    
    float ms = endEvent(&startEvent, &stopEvent);
    
    #ifndef MEASURES
        // Error tracking and event time
        gpuErrchk( hipPeekAtLastError() );
        // Calculate theoretical occupancy
        int maxActiveBlocks;
        hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, cosKernel, blockSize, 0);

        int device;
        hipDeviceProp_t props;
        hipGetDevice(&device);
        hipGetDeviceProperties(&props, device);
        
        float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / 
                        (float)(props.maxThreadsPerMultiProcessor / 
                                props.warpSize);

        std::cout << "Launched blockSize: " << BLOCK<< std::endl
                << "Min Grid Size: " << minGridSize << std::endl
                << "Launched Grid Size: " << GRID << std::endl
                << "Max active blocks: " << maxActiveBlocks<< std::endl
                << "Theoretical occupancy:" << occupancy << std::endl;
   #endif           
  
    return ms;
}
#endif

/**** STREAM ****/
#ifdef STREAM
void streamCosine(int m, int chunk, float *x, float *cosx, float *x_d, hipStream_t strm, int strBytes)
{     
    // H2D mem copy 
    gpuErrchk( hipMemcpyAsync(x_d, x, strBytes, hipMemcpyHostToDevice, strm) ); 
    // Kernel call
    cosKernel<<<GRID, BLOCK, 0, strm>>>(m, chunk, x_d); 
    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
    #endif   
    // D2H mem copy 
    gpuErrchk( hipMemcpyAsync( cosx, x_d, strBytes, hipMemcpyDeviceToHost, strm) );
}


void cosine(int m, int chunk, float *x, float *cosx, float *x_d)
{   
    int xBytes = chunk*sizeof(float);
    // H2D mem copy 
    gpuErrchk( hipMemcpy(x_d, x, xBytes, hipMemcpyHostToDevice) ); 
    // Kernel call
    cosKernel<<<GRID, BLOCK>>>(m, chunk, x_d);  
    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
    #endif   
    // D2H mem copy 
    gpuErrchk( hipMemcpy( cosx, x_d, xBytes, hipMemcpyDeviceToHost) );
}
#endif


/**** FUTURE ****/
#ifdef FUTURE
std::vector<std::future<float *>> 
    streamFutureCosine(int M, int chunk, float *cosx, float *x, float *x_d, hipStream_t *streams, int nStreams)
{
    std::vector<std::future<float *>> futures; 

    for(int i = 0; i < N_size/chunk; ++i) {
        const int k = i%nStreams;
        randomArray(x+i*chunk,chunk);

        const int strOffs = k*chunk;
        float *p_xd = x_d+strOffs;
        float *p_x = x+(i*chunk);
        float *p_cosx = cosx+(i*chunk);

        futures.push_back (std::async(std::launch::async,     
            [=] (hipStream_t strm, int strBytes) {
            // H2D mem copy 
            hipMemcpyAsync(p_xd, p_x, strBytes, hipMemcpyHostToDevice,strm);        
            // Kernel call
            cosKernel<<<GRID, BLOCK,0,strm>>>(M, chunk, p_xd); 
            // D2H mem copy 
            hipMemcpyAsync(p_cosx, p_xd, strBytes, hipMemcpyDeviceToHost,strm) ; 
            #ifndef MEASURES
                gpuErrchk( hipPeekAtLastError() );
            #endif 

            return p_cosx;
        }, streams[k], chunk*sizeof(float) ));       
    }
    return futures;
}
#endif


/**** STREAM MANAGED ****/
#ifdef MANAGED
void  unifiedStreamCosine(int m, int chunk, float *x, float *cosx, hipStream_t strm)
{
    randomArray(x,chunk);
    memcpy(cosx,x,chunk);

    #ifdef LOWPAR
        cosGridStride<<<GRID, BLOCK, 0, strm>>>(m, chunk, cosx);
    #else
        cosKernel<<<GRID, BLOCK, 0, strm>>>(m, chunk, cosx);
    #endif

    hipStreamSynchronize(strm);
}
#endif