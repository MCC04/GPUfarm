#include "hip/hip_runtime.h"
//#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
 
#include <cstdlib>
#include <algorithm>
#include <ctime>
#include <vector>
#include <future>
#include <iterator>
//#include <cudaUtils.h>
#include <cosFutStr.h>

#define HIGH 500.0f
#define LOW -500.0f



void randomArray(float *x, int n){
    #ifndef MEASURES
            std::cout<<std::endl<< "X ARRAY: "<<std::endl;  
    #endif
    for(int i=0; i<n;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
        #ifndef MEASURES
            std::cout<< x[i] << ", ";  
        #endif
    }
}

/*float getMatrixVal(float *mat, int row, int col, int width)
{
    return mat[row + col*width];
}

void setMatrixVal(float *mat, int row, int col, int width, float val)
{   
    mat[row + col*width] = val;
}

/*void randomMatrix(const int m, int n,float *mat){
    #ifndef MEASURES
        std::cout<< "MATRIX M: "<<std::endl;  
    #endif

    for(int r = 0; r<m; ++r){
        for(int c = 0; c<n; ++c){
            float val=LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;
            setMatrixVal(mat, r, c, n, val);

            #ifndef MEASURES
                std::cout<< getMatrixVal(mat,r,c,n) << ", ";              
            #endif
        }
        #ifndef MEASURES
            std::cout<< std::endl;  
        #endif
    }       
}

int* getGaussian(int height, int width, double sigma)
{
    //Matrix kernel(height, Array(width));
    double sum=0.0;
    int i,j;

    for (i=0 ; i<height ; i++) {
        for (j=0 ; j<width ; j++) {
            kernel[i][j] = exp(-(i*i+j*j)/(2*sigma*sigma))/(2*M_PI*sigma*sigma);
            sum += kernel[i][j];
        }
    }

    for (i=0 ; i<height ; i++) {
        for (j=0 ; j<width ; j++) {
            kernel[i][j] /= sum;
        }
    }

    return kernel;
}*/

/*********
**KERNELS*
**********/
__global__ void emptyKernel(){ return; }

__global__ void cosKernel(int M, int N, float *x_d, int *myclocks, int offset){    
    int idx = offset+blockIdx.x*blockDim.x + threadIdx.x; 
   
    clock_t start =clock();

    for(int j=0;j<M;j+=1)
        x_d[idx]=cosf(x_d[idx]);  

    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);
    return ;
}

//M = iterations; N = size
__global__ void cosGridStride(int M, int N, float *x_d, int offset, int *myclocks){    
    int index = offset+blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    clock_t start =clock();
    for (int i = index; i < N; i += stride)
    {
        for(int j=0;j<M;j+=1)
            x_d[i]=cosf(x_d[i]);  
    }
    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);

    return ;
}

/*__global__ void matMulKernel(float* Ad, float* Bd, float* Cd, int m, int n, int k)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if(row<m && col<n) {
        float sum = 0;
        for(int j=0;j<k;j++) {
            sum += Ad[row*k+j] * Bd[j*n+col];
        }
        Cd[row*n+col] = sum;
    }
}

__global__
void blurKernel(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 5; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
        }
}*/


//KERNEL LAUNCERS
float emptyKer(){
    float ms=0;
    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );   

    checkCuda( hipEventRecord(startEvent,0) );
    
    emptyKernel<<<GRID, BLOCK>>>();

    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(ms, startEvent, stopEvent) );

    return ms;
    
}

void cosKer(std::vector<my_struct> &getDatas,int bytesSize )
{
    std::vector<std::future<my_struct>> futures;
    int *clocks_d;
    float *x_d;    

    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );    

    checkCuda(hipMalloc(&x_d, bytesSize)); 
    checkCuda(hipMalloc(&clocks_d, GRID*sizeof(int)));

    for(int i = 0; i < K_exec; ++i) {
        futures.push_back (std::async(std::launch::deferred,
            [&]() { 
                my_struct _xs;
                _xs.clocks=new int[GRID];
                _xs.x_vect=new float[N_size];
                randomArray(_xs.x_vect, N_size);

                checkCuda( hipEventRecord(startEvent,0) );

                checkCuda(hipMemcpy(x_d, _xs.x_vect, bytesSize, hipMemcpyHostToDevice)); 

                #ifdef STRIDE
                    cosGridStride<<<GRID, BLOCK>>>(M_iter, N_size, x_d, 0, clocks_d);
                #else
                    cosKernel<<<GRID, BLOCK>>>(M_iter, N_size, x_d,clocks_d, 0);
                #endif
                              
                checkCuda(hipMemcpy( _xs.x_vect, x_d, bytesSize, hipMemcpyDeviceToHost));
                checkCuda(hipMemcpy(_xs.clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost));

                checkCuda( hipEventRecord(stopEvent, 0) );
                checkCuda( hipEventSynchronize(stopEvent) );
                checkCuda( hipEventElapsedTime(&_xs.eventTime, startEvent, stopEvent) );

                return _xs;
            }));          
    }
    for(auto &e : futures) 
        getDatas.push_back(e.get());
}



void cosKerStream(
    int m, int n,
    float *x, //float *cosx,  
    int *clocks, 
    int offset, hipStream_t strm)
{
        #ifdef STRIDE
            cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, n, x, clocks, offset);
        #else
            cosKernel<<<GRID, BLOCK, offset, strm>>>(m, n, x, clocks, offset);
        #endif
}




float  cosKerStream(
    hipEvent_t start, hipEvent_t stop,
    int m, int n,
    float *x, float *cosx,  int *clocks, 
    int offset, hipStream_t strm)
{
    float ms;  
    randomArray(x, n);
    memcpy(cosx,x,N_size);
    checkCuda( hipEventRecord(start,0) );

    #ifdef STRIDE
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
    #else
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
     
    return ms;
}
/*
float matMulKer(
    float *Ad, float *Bd, float *Cd, 
    int m, int k, int n, 
    hipStream_t strm, hipEvent_t start, hipEvent_t stop)
{

    float ms;
    #ifdef LOWPAR
        dim3 dimBlock(32,32);
        dim3 dimGrid(1,1); 
    #else
        dim3 dimBlock(BLOCK,BLOCK);
        dim3 dimGrid((m+dimBlock.x-1)/dimBlock.x, (n+dimBlock.y-1)/dimBlock.y); 
    #endif
    

    randomMatrix(m,k, Ad);
    randomMatrix(k,n, Bd); 

    checkCuda( hipEventRecord(start,0) );

    matMulKernel<<<dimGrid, dimBlock, 0, strm>>>(Ad, Bd, Cd, m,  k,  n);

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );


    return ms;
}

float filter (
    unsigned char *img_in, unsigned char *img_out,
    int width, int height,
    hipStream_t strm,
    hipEvent_t start, hipEvent_t stop)
{    
    float ms=0;
    int bytes=width*height*3*sizeof(unsigned char);
       
    dim3 blockDims(512,1,1);
    dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );

    checkCuda( hipEventRecord(start,0) ); 

    hipMallocManaged(&img_in, bytes);
    hipMallocManaged(&img_out, bytes);

    blurKernel<<<gridDims, blockDims, 0, strm>>>(img_in, img_out, width, height); 

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
 
    return ms;
}
*/