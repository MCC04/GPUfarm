#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h> 
#include <algorithm>

#include <cosFutStr.h>

#define HIGH 500.0f
#define LOW -500.0f

void randomArray(float *x, int n){
    for(int i=0; i<n;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
    }
}

/*********
**KERNELS*
**********/
#ifdef EMPTY
__global__ void emptyKernel(){ return; }
#endif

/**** GRID-STRIDE COS KERNEL ****/ 
__global__ void cosKernel(int M, int N, float *x_d, int *myclocks, int offset){    
    int idx = offset+blockIdx.x*blockDim.x + threadIdx.x; 
   
    if(idx<N){
        clock_t start =clock();

        for(int j=0; j<M; ++j)
            x_d[idx]=cosf(x_d[idx]);  

        clock_t end=clock();

        if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);
    }
    return ;
}


/**** GRID-STRIDE COS KERNEL ****/ 
__global__ void cosGridStride(int M, int N, float *x_d, int *myclocks, int offset){    
    int index = offset+blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    clock_t start =clock();
    for (int i = index; i < N; i += stride)
    {
        for(int j=0;j<M;j+=1)
            x_d[i]=cosf(x_d[i]);  
    }
    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);

    return ;
}


/******************
* KERNEL LAUNCERS *
*******************/

/**** FUTURE ****/
#ifdef FUTURE
std::vector<std::future<hostData_t>> 
    cosKerFuture(int M, int chunk, hostData_t output, float *x, float *x_d, int *clocks_d, hipStream_t *strm, int nStreams, int offset)
{
    std::vector<std::future<hostData_t>> futures; 

    for(int i = 0; i < K_exec; ++i) {
        const int k = i%nStreams;
        randomArray(x+i*chunk,chunk);

        const int strOffs = k*chunk;
        float *p_xd = x_d+strOffs;
        float *p_x = x+(i*chunk);
        int *p_clocks= clocks_d+(k*GRID);

        futures.push_back (std::async(std::launch::async,     
            [=] (hipStream_t strm, int strBytes) {
            //[M, chunk, p_xd, p_clocks, output, offset,i]

            //std::cout <<i<<"- going to memcpy x in H2D..."<<std::endl;        
            hipMemcpy(p_xd, p_x, strBytes, hipMemcpyHostToDevice);        
            //std::cout <<i<<"- done memcpy x in H2D!"<<std::endl;

            #ifdef LOWPAR
                //std::cout <<i<<"- kernel launch..."<<std::endl;
                cosGridStride<<<GRID, BLOCK,0,strm>>>(M, chunk, p_xd, p_clocks, offset); 
                //std::cout <<i<<"- kernel end!"<<std::endl;
            #else
                //  std::cout <<i<<"- kernel launch..."<<std::endl;
                cosKernel<<<GRID, BLOCK,0,strm>>>(M, chunk, p_xd, p_clocks, offset); 
                //std::cout <<i<<"- kernel end!"<<std::endl;
            #endif

            //std::cout <<i<<"- going to memcpy x in D2H..."<<std::endl;
            hipMemcpy(output.x, p_xd, strBytes, hipMemcpyDeviceToHost) ; 
            hipMemcpy(output.clocks, p_clocks, GRID*sizeof(int), hipMemcpyDeviceToHost) ; 
            //std::cout <<i<<"- done memcpy x in D2H!"<<std::endl;

            return output;
        }, strm[k], chunk*sizeof(float) ));       


        /*futures.push_back (std::async(std::launch::async,//std::launch::deferred,//       
            [M, chunk, x_d, clocks_d, output, offset,i] (float * x, hipStream_t strm, int strBytes) {
            
            //std::cout <<i<<"- going to memcpy x in H2D..."<<std::endl;
            
            
            gpuErrchk( hipMemcpyAsync(x_d, x, strBytes, hipMemcpyHostToDevice, strm) ); //gpuErrchk( hipMemcpy(&x_d[i*chunk], &x[i*chunk], chunk*sizeof(float), hipMemcpyHostToDevice) );          

            //std::cout <<i<<"- done memcpy x in H2D!"<<std::endl;



            #ifdef LOWPAR
                //std::cout <<i<<"- kernel launch..."<<std::endl;

                cosGridStride<<<GRID, BLOCK,0,strm>>>(M, chunk, x_d, clocks_d, offset); //cosGridStride<<<GRID, BLOCK>>>(M_iter, chunk, &x_d[i*chunk], &clocks_d[i*chunk], 0);

                //std::cout <<i<<"- kernel end!"<<std::endl;
            #else

                //  std::cout <<i<<"- kernel launch..."<<std::endl;

                cosKernel<<<GRID, BLOCK,0,strm>>>(M, chunk, x_d, clocks_d, offset); //cosKernel<<<GRID, BLOCK>>>(M_iter, chunk, &x_d[i*chunk],&clocks_d[i*chunk], 0);

                //std::cout <<i<<"- kernel end!"<<std::endl;

            #endif

            //std::cout <<i<<"- going to memcpy x in D2H..."<<std::endl;

            gpuErrchk( hipMemcpyAsync(output.x, x_d, strBytes, hipMemcpyDeviceToHost, strm) ); //gpuErrchk( hipMemcpy( output.x_vect, &x_d[i*chunk], chunk*sizeof(float), hipMemcpyDeviceToHost) );
            gpuErrchk( hipMemcpyAsync(output.clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost, strm) ); //gpuErrchk( hipMemcpy(output.clocks, &clocks_d[i*GRID], GRID*sizeof(int), hipMemcpyDeviceToHost) );

            hipStreamSynchronize(strm);
            //std::cout <<i<<"- done memcpy x in D2H!"<<std::endl;

            return output;
        }, x+(i*chunk), strm[k], chunk*sizeof(float) ));*/  
    }
    return futures;
}
#endif


/**** STREAM ****/
#ifdef STREAM

void cosKerStream(int m, int chunk, float *x, float *cosx, float *x_d, int *clocks, int *clocks_d, hipStream_t strm, int strBytes, int offset)
{    
    gpuErrchk( hipMemcpyAsync(x_d, x, strBytes, hipMemcpyHostToDevice, strm) ); 
    #ifdef LOWPAR
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, x_d, clocks_d, offset);
    #else
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, x_d, clocks_d, offset);
    #endif   

    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    #endif   
    gpuErrchk( hipMemcpyAsync( cosx, x_d, strBytes, hipMemcpyDeviceToHost, strm) );
    gpuErrchk( hipMemcpyAsync( clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost, strm) );

    #ifndef MEASURES
        printClocks(clocks,GRID);
    #endif  
    //hipStreamSynchronize(strm);       
}
#endif


/**** STREAM MANAGED ****/
#ifdef MANAGED
void  cosKerStream(int m, int chunk,//int n,
    float *x, float *cosx,  int *clocks, int offset, hipStream_t strm)
{
    randomArray(x,chunk);
    memcpy(cosx,x,chunk);

    #ifdef LOWPAR
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #else
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #endif

    hipStreamSynchronize(strm);
}
#endif


/**** EMPTY ****/
#ifdef EMPTY
float emptyKer(){
    float ms=0;
    hipEvent_t startEvent, stopEvent;
    gpuErrchk( hipEventCreate(&startEvent) );
    gpuErrchk( hipEventCreate(&stopEvent) );   

    gpuErrchk( hipEventRecord(startEvent,0) );
    
    emptyKernel<<<GRID, BLOCK>>>();

    gpuErrchk( hipEventRecord(stopEvent, 0) );
    gpuErrchk( hipEventSynchronize(stopEvent) );
    gpuErrchk( hipEventElapsedTime(&ms, startEvent, stopEvent) );

    return ms;    
}
#endif