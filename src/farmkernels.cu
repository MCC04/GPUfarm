#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
 
#include <cstdlib>
#include <algorithm>
#include <ctime>
#include <vector>
#include <future>
#include <iterator>
#include <cosFutStr.h>

#define HIGH 500.0f
#define LOW -500.0f


void randomArray(float *x, int n){
    /*#ifndef MEASURES
            std::cout<<std::endl<< "X ARRAY: "<<std::endl;  
    #endif*/
    for(int i=0; i<n;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
       /* #ifndef MEASURES
            std::cout<< x[i] << ", ";  
        #endif*/
    }
}

/*********
**KERNELS*
**********/
#ifdef EMPTY
__global__ void emptyKernel(){ return; }
#endif

/**** GRID-STRIDE COS KERNEL ****/ 
__global__ void cosKernel(int M, int N, float *x_d, int *myclocks, int offset){    
    int idx = offset+blockIdx.x*blockDim.x + threadIdx.x; 
   
    if(idx<N){
        clock_t start =clock();

        for(int j=0; j<M; ++j)
            x_d[idx]=cosf(x_d[idx]);  

        clock_t end=clock();

        if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);
    }
    return ;
}


/**** GRID-STRIDE COS KERNEL ****/ 
__global__ void cosGridStride(int M, int N, float *x_d, int *myclocks, int offset){    
    int index = offset+blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    clock_t start =clock();
    for (int i = index; i < N; i += stride)
    {
        for(int j=0;j<M;j+=1)
            x_d[i]=cosf(x_d[i]);  
    }
    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);

    return ;
}


/******************
* KERNEL LAUNCERS *
*******************/

/**** STREAM ****/
#ifdef FUTURE
float cosKer(std::vector<my_struct> &getDatas, int chunk, int bytesSize )
{
    std::vector<std::future<my_struct>> futures;
    int *clocks_d;
    float *x_d; 

    float *x = new float[N_size]; //gpuErrchk( hipHostMalloc((void **)&x, N_size*sizeof(float)) ); //pinned x    
    //float *cosx = new float[N_size]; //gpuErrchk( hipHostMalloc((void **)&cosx, N_size*sizeof(float)) ); //pinned x    
    float *clockss = new float[K_exec*GRID]; //gpuErrchk( hipHostMalloc((void **)&clockss, K_exec*GRID*sizeof(float)) ); //pinned x
  

    gpuErrchk( hipMalloc((void **)&x_d, N_size*sizeof(float)) ); 
    gpuErrchk( hipMalloc((void **)&clocks_d, GRID*K_exec*sizeof(int)) );

    hipEvent_t startEvent, stopEvent;
    createAndStartEvent(startEvent, stopEvent);
    /*gpuErrchk( hipEventCreate(&startEvent) );
    gpuErrchk( hipEventCreate(&stopEvent) ); 
    gpuErrchk( hipEventRecord(startEvent,0) );*/

    randomArray(x, N_size);

    hipStream_t *stream=streamCreate(3);
    for(int i = 0; i < K_exec; ++i) {
        
     
        futures.push_back (
         std::async(std::launch::async,//std::launch::deferred,//       
             [x,x_d,clocks_d,chunk,stream](int i ) { //[x,x_d,clocks_d,chunk](int i ) { 

                my_struct _xs;
                _xs.clocks=new int[GRID];
                _xs.x_vect=new float[chunk];
                int k = i%3;

                #ifndef MEASURES
                    std::cout <<i<<" - going to memcpy x in H2D..."<<std::endl;
                #endif                
                gpuErrchk( hipMemcpyAsync(&x_d[i*chunk], &x[i*chunk], chunk*sizeof(float), hipMemcpyHostToDevice, stream[k]) ); //gpuErrchk( hipMemcpy(&x_d[i*chunk], &x[i*chunk], chunk*sizeof(float), hipMemcpyHostToDevice) );          
                #ifndef MEASURES
                    std::cout << i<<"- done memcpy x in H2D!"<<std::endl;
                #endif

                #ifdef LOWPAR
                    #ifndef MEASURES                    
                        std::cout << i<<"- kernel launch..."<<std::endl;
                    #endif
                    cosGridStride<<<GRID, BLOCK,0,stream[k]>>>(M_iter, chunk, &x_d[i*chunk], &clocks_d[i*chunk], 0); //cosGridStride<<<GRID, BLOCK>>>(M_iter, chunk, &x_d[i*chunk], &clocks_d[i*chunk], 0);
                    #ifndef MEASURES 
                        std::cout << i<<"- kernel end!"<<std::endl;
                    #endif
                #else
                   
                 
                    #ifndef MEASURES                    
                        std::cout << i<<"- kernel launch..."<<std::endl;
                    #endif
                    cosKernel<<<GRID, BLOCK,0,stream[k]>>>(M_iter, chunk, &x_d[i*chunk],&clocks_d[i*chunk], 0); //cosKernel<<<GRID, BLOCK>>>(M_iter, chunk, &x_d[i*chunk],&clocks_d[i*chunk], 0);
                    #ifndef MEASURES 
                        std::cout << i<<"- kernel end!"<<std::endl;
                    #endif

                #endif
                              
                #ifndef MEASURES
                    std::cout <<i<<" - going to memcpy x in D2H..."<<std::endl;
                #endif  
                gpuErrchk( hipMemcpyAsync(_xs.x_vect, &x_d[i*chunk], chunk*sizeof(float), hipMemcpyDeviceToHost, stream[k]) ); //gpuErrchk( hipMemcpy( _xs.x_vect, &x_d[i*chunk], chunk*sizeof(float), hipMemcpyDeviceToHost) );
                gpuErrchk( hipMemcpyAsync(_xs.clocks, &clocks_d[i*GRID], GRID*sizeof(int), hipMemcpyDeviceToHost, stream[k]) ); //gpuErrchk( hipMemcpy(_xs.clocks, &clocks_d[i*GRID], GRID*sizeof(int), hipMemcpyDeviceToHost) );
                #ifndef MEASURES
                    std::cout << i<<"- done memcpy x in D2H!"<<std::endl;
                #endif
                _xs.eventTime=0;

                return _xs;
            },i));       
    }
    float ms=0.0f;

    
    for(auto &e : futures) 
            getDatas.push_back(e.get()); 
    
    /*gpuErrchk( hipEventRecord(stopEvent, 0) );
    gpuErrchk( hipEventSynchronize(stopEvent) );
    gpuErrchk( hipEventElapsedTime(&ms, startEvent, stopEvent) );*/
    msTot = endEvent(startEvent, stopEvent);

    #ifndef MEASURES
        std::cout << "EVENT TIME FUTURE: "<< ms<<std::endl;
    #endif
    streamDestroy(stream,3);

    hipHostFree(x);
    //hipHostFree(cosx);
    hipHostFree(clockss);
    hipFree(x_d);
    hipFree(clocks_d);

    return ms;
}
#endif

/**** STREAM ****/
#ifdef STREAM

void cosKerStream(int m, int chunk, float *x, float *cosx, float *x_d, int *clocks, int *clocks_d, hipStream_t strm, int strBytes, int offset)
{
    
    gpuErrchk( hipMemcpyAsync(x_d, x, strBytes, hipMemcpyHostToDevice, strm) ); 
    #ifdef LOWPAR
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, x_d, clocks_d, offset);
    #else
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, x_d, clocks_d, offset);
    #endif   

    #ifndef MEASURES
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    #endif   
    gpuErrchk( hipMemcpyAsync( cosx, x_d, strBytes, hipMemcpyDeviceToHost, strm) );
    gpuErrchk( hipMemcpyAsync( clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost, strm) );

    #ifndef MEASURES
        printClocks(clocks,GRID);
    #endif      
    
}
#endif


/**** STREAM MANAGED ****/
#ifdef MANAGED
void  cosKerStream(
    //hipEvent_t start, hipEvent_t stop,
    int m, int chunk,//int n,
    float *x, float *cosx,  int *clocks, 
    int offset, hipStream_t strm)
{
    float ms;  
    //randomArray(x, n);
    //memcpy(cosx,x,N_size);
    randomArray(x,chunk);
    memcpy(cosx,x,chunk);
    
    //gpuErrchk( hipEventRecord(start,0) );

    //#ifdef STRIDE
    #ifdef LOWPAR
        //cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #else
        //cosKernel<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, chunk, cosx, clocks, offset);
    #endif

    /*gpuErrchk( hipEventRecord(stop, 0) );
    gpuErrchk( hipEventSynchronize(stop) );
    gpuErrchk( hipEventElapsedTime(&ms, start, stop) );*/
     
   // return ms;
}
#endif


/**** EMPTY ****/
#ifdef EMPTY
float emptyKer(){
    float ms=0;
    hipEvent_t startEvent, stopEvent;
    gpuErrchk( hipEventCreate(&startEvent) );
    gpuErrchk( hipEventCreate(&stopEvent) );   

    gpuErrchk( hipEventRecord(startEvent,0) );
    
    emptyKernel<<<GRID, BLOCK>>>();

    gpuErrchk( hipEventRecord(stopEvent, 0) );
    gpuErrchk( hipEventSynchronize(stopEvent) );
    gpuErrchk( hipEventElapsedTime(&ms, startEvent, stopEvent) );

    return ms;    
}
#endif