#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
 
#include <cstdlib>
#include <algorithm>
#include <ctime>
#include <vector>
#include <future>
#include <iterator>
#include <cosFutStr.h>

#define HIGH 500.0f
#define LOW -500.0f


void randomArray(float *x, int n){
    #ifndef MEASURES
            std::cout<<std::endl<< "X ARRAY: "<<std::endl;  
    #endif
    for(int i=0; i<n;i+=1){
        x[i] = LOW + (float) std::rand() * (HIGH-LOW) / RAND_MAX;   
        #ifndef MEASURES
            std::cout<< x[i] << ", ";  
        #endif
    }
}

/*********
**KERNELS*
**********/
__global__ void emptyKernel(){ return; }

__global__ void cosKernel(int M, int N, float *x_d, int *myclocks, int offset){    
    int idx = offset+blockIdx.x*blockDim.x + threadIdx.x; 
   
    clock_t start =clock();

    for(int j=0;j<M;j+=1)
        x_d[idx]=cosf(x_d[idx]);  

    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);
    return ;
}

__global__ void cosGridStride(int M, int N, float *x_d, int offset, int *myclocks){    
    int index = offset+blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    clock_t start =clock();
    for (int i = index; i < N; i += stride)
    {
        for(int j=0;j<M;j+=1)
            x_d[i]=cosf(x_d[i]);  
    }
    clock_t end=clock();

    if (threadIdx.x == 0) myclocks[blockIdx.x+(offset/blockDim.x)]=(int)(end-start);

    return ;
}


//KERNEL LAUNCERS
float emptyKer(){
    float ms=0;
    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );   

    checkCuda( hipEventRecord(startEvent,0) );
    
    emptyKernel<<<GRID, BLOCK>>>();

    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(ms, startEvent, stopEvent) );

    return ms;    
}

void cosKer(std::vector<my_struct> &getDatas,int bytesSize )
{
    std::vector<std::future<my_struct>> futures;
    int *clocks_d;
    float *x_d;    

    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );    

    checkCuda(hipMalloc(&x_d, bytesSize)); 
    checkCuda(hipMalloc(&clocks_d, GRID*sizeof(int)));

    for(int i = 0; i < K_exec; ++i) {
        futures.push_back (std::async(std::launch::deferred,
            [&]() { 
                my_struct _xs;
                _xs.clocks=new int[GRID];
                _xs.x_vect=new float[N_size];
                randomArray(_xs.x_vect, N_size);

                checkCuda( hipEventRecord(startEvent,0) );

                checkCuda(hipMemcpy(x_d, _xs.x_vect, bytesSize, hipMemcpyHostToDevice)); 

                #ifdef STRIDE
                    cosGridStride<<<GRID, BLOCK>>>(M_iter, N_size, x_d, 0, clocks_d);
                #else
                    cosKernel<<<GRID, BLOCK>>>(M_iter, N_size, x_d,clocks_d, 0);
                #endif
                              
                checkCuda(hipMemcpy( _xs.x_vect, x_d, bytesSize, hipMemcpyDeviceToHost));
                checkCuda(hipMemcpy(_xs.clocks, clocks_d, GRID*sizeof(int), hipMemcpyDeviceToHost));

                checkCuda( hipEventRecord(stopEvent, 0) );
                checkCuda( hipEventSynchronize(stopEvent) );
                checkCuda( hipEventElapsedTime(&_xs.eventTime, startEvent, stopEvent) );

                return _xs;
            }));          
    }
    for(auto &e : futures) 
        getDatas.push_back(e.get());
}

void cosKerStream(
    int m, int n,
    float *x, int *clocks, 
    int offset, hipStream_t strm)
{
        #ifdef STRIDE
            cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, n, x, clocks, offset);
        #else
            cosKernel<<<GRID, BLOCK, offset, strm>>>(m, n, x, clocks, offset);
        #endif
}

float  cosKerStream(
    hipEvent_t start, hipEvent_t stop,
    int m, int n,
    float *x, float *cosx,  int *clocks, 
    int offset, hipStream_t strm)
{
    float ms;  
    randomArray(x, n);
    memcpy(cosx,x,N_size);
    checkCuda( hipEventRecord(start,0) );

    #ifdef STRIDE
        cosGridStride<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
    #else
        cosKernel<<<GRID, BLOCK, offset, strm>>>(m, n, cosx, clocks, offset);
    #endif

    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&ms, start, stop) );
     
    return ms;
}
