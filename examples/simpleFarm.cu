#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <ctime>
#include <unistd.h>

//workers computing square of rands
__global__ void kerSquare(int *randsDev,int* resDev){
    int myId = blockIdx.x * blockDim.x + threadIdx.x;

    //std::cout << myId << ", ";

    resDev[myId] = randsDev[myId] * randsDev[myId];
}

int main(){
    std::srand(std::time(NULL));
    int n = 20000;
    int arraySize=2000;
    int size = arraySize*sizeof(int);
    int *randsDev, *resDev, tmp[arraySize], finalRes[n], offs;
    int count;
    int random_variable;
    unsigned int microseconds = 400;

    if(microseconds<=400){
        hipMalloc(&resDev, size);
        dim3 grid(10,1);
        dim3 block(200,1);

        for(int i=0;i<n;i+=1){
            //emitter
            random_variable = std::rand()%100;
            tmp[count]=random_variable;

            count+=1;
            usleep(microseconds);
            if(count==arraySize){
                hipMalloc(&randsDev, size);
                std::cout << "copying randoms to device mem"<< std::endl;
                hipMemcpy(randsDev, tmp, size, hipMemcpyHostToDevice);

                //worker
                std::cout << "calling ker function"<< std::endl;
                kerSquare<<<grid,block>>>(randsDev, resDev);

                //collector
                hipMemcpy(tmp, resDev, size, hipMemcpyDeviceToHost);
                std::cout << std::endl << "copying back results"<< std::endl;
                offs+=count;
                std::copy(tmp, tmp+arraySize-1, finalRes+offs);

                count=0;
            }
        }
    }
    else{
        //Execute on CPU?
    }

    for(int i=0;i<n;i+=1){
        std::cout<<"\t"<<finalRes[i];
    }

    std::cout<<std::endl;
    //free mem
    hipFree(randsDev);
    hipFree(resDev);
}