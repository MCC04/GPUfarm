#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void kerArraySum(int *md,int* nd, int *pd){
    //int myId=threadIdx.x;

    int myId = blockIdx.x * blockDim.x + threadIdx.x;

    p[myId] = m[myId] + n[myId];
}

int main(){
    int size = 2000*sizeof(int);
    int m[2000],n[2000],p[2000],nd,md,pd;

    //arrays init
    for(int i=0;i<2000;i+=1){
        m[i]=i;
        n[i]=i;
    }

    //mem allocation and copy
    hipMalloc((void**)&md, size);
    hipMemcpy(md, m, size, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&nd, size);
    hipMemcpy(nd, nd, size, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&pd, size);
    
    //setting grid and block dims
    dim3 grid(10,1);
    dim3 block(200,1);

    //calling the kernel funct
    kerArraySum<<<grid,block>>>(md,nd,pd,size);

    //copy results back to cpu mem
    hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);

    for(int i=0;i<2000;i+=1){
        std::cout<<"\t"<<p[i];
    }
    std::cout<<std::endl;

    //free mem
    hipFree(md);
    hipFree(nd);
    hipFree(pd);
}